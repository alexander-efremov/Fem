#include "hip/hip_runtime.h"
#include "common.h"
#include "point.h"
#include "utils.h"
#include "compute_density_cuda.cuh"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hemi.h>
__constant__ double C_B;
__constant__ double C_LB;
__constant__ double C_RB;
__constant__ double C_UB;
__constant__ double C_BB;
__constant__ double C_INVERTED_HX_HY;
__constant__ double C_HX;
__constant__ double C_HY;
__constant__ int C_OY_LEN;
__constant__ int C_OX_LEN;
__constant__ int C_OX_LEN_1;
__constant__ int C_XY_LEN;
__constant__ double C_PREV_TIME; // tau * (tl - 1)
__constant__ double C_TIME;
__constant__ double C_TAU;
__device__ double *OX_DEVICE, *OY_DEVICE;

#define sqr(x) ((x)*(x))
#define cub(x) ((x)*(x)*(x))
#define quad(x) ((x)*(x)*(x)*(x))

#ifdef __NVCC__
#define __pure __device__
#endif

static double B; //-V707
static double UB; //-V707
static double BB; //-V707
static double LB; //-V707
static double RB; //-V707
static double TAU;
static int OX_LEN;
static int OX_LEN_1; // OX_LEN_1
static int OY_LEN;
static int XY_LEN;
static int TIME_STEP_CNT;
static double HX; //-V707
static double HY; //-V707
static double* OX; //-V707
static double* OY; //-V707
static double TIME;
static double INVERTED_HX_HY;

__pure inline static void sort_by_y_asc(c_dp_t& x, c_dp_t& y, c_dp_t& z)
{	
	double t;
	if (x.y < y.y)
	{
		if (z.y < x.y) 
		{
			//swap(x, z);
			double t = x.x;
			x.x = z.x;
			z.x = t;
			t = x.y;
			x.y = z.y;
			z.y = t;
		}
	}
	else
	{
		if (y.y < z.y) 
		{
			//swap(x, y);
			t = x.x;
			x.x = y.x;
			y.x = t;
			t = x.y;
			x.y = y.y;
			y.y = t;
		}
		else 
		{
			//swap(x, z);
			t = x.x;
			x.x = z.x;
			z.x = t;
			t = x.y;
			x.y = z.y;
			z.y = t;
		}
	}
	if (z.y < y.y) 
	{
		//swap(y, z);
		t = y.x;
		y.x = z.x;
		z.x = t;
		t = y.y;
		y.y = z.y;
		z.y = t;
	}
}

__pure inline void sort_by_y(c_dp_t* a)
{
	for (int i = 1; i < 4; i++)
	{
		for (int j = i; j > 0 && a[j - 1].y > a[j].y; j--)
		{
			double t = a[j].x;
			a[j].x = a[j - 1].x;
			a[j - 1].x = t;
			t = a[j].y;
			a[j].y = a[j - 1].y;
			a[j - 1].y = t;
		}
	}
}

__pure inline void sort_by_y_desc_3(c_dp4_t* a)
{
	for (int i = 2; i < 4; i++)
	{
		for (int j = i; j > 1 && a[j - 1].y < a[j].y; j--)
		{
			double t = a[j].x;
			a[j].x = a[j - 1].x;
			a[j - 1].x = t;
			t = a[j].y;
			a[j].y = a[j - 1].y;
			a[j - 1].y = t;
			t = a[j].x_initial;
			a[j].x_initial = a[j - 1].x_initial;
			a[j - 1].x_initial = t;
			t = a[j].y_initial;
			a[j].y_initial = a[j - 1].y_initial;
			a[j - 1].y_initial = t;
		}
	}
}

__pure inline void sort_by_x_asc(c_dp4_t* a)
{
	for (int i = 1; i < 4; i++)
	{
		for (int j = i; j > 0 && a[j - 1].x > a[j].x; j--)
		{
			double t = a[j].x;
			a[j].x = a[j - 1].x;
			a[j - 1].x = t;
			t = a[j].y;
			a[j].y = a[j - 1].y;
			a[j - 1].y = t;
			t = a[j].x_initial;
			a[j].x_initial = a[j - 1].x_initial;
			a[j - 1].x_initial = t;
			t = a[j].y_initial;
			a[j].y_initial = a[j - 1].y_initial;
			a[j - 1].y_initial = t;
		}
	}

	if (a[0].y > a[1].y)
	{
		double t = a[0].x;
		a[0].x = a[1].x;
		a[1].x = t;
		t = a[0].y;
		a[0].y = a[1].y;
		a[1].y = t;
		t = a[0].x_initial;
		a[0].x_initial = a[1].x_initial;
		a[1].x_initial = t;
		t = a[0].y_initial;
		a[0].y_initial = a[1].y_initial;
		a[1].y_initial = t;
	}
	if (a[2].y < a[3].y)
	{
		double t = a[2].x;
		a[2].x = a[3].x;
		a[3].x = t;
		t = a[2].y;
		a[2].y = a[3].y;
		a[3].y = t;

		t = a[2].x_initial;
		a[2].x_initial = a[3].x_initial;
		a[3].x_initial = t;
		t = a[2].x_initial;
		a[2].x_initial = a[3].x_initial;
		a[3].x_initial = t;
	}
}

// ïîëó÷àåòñÿ ïîðÿäîê
/*

a[1]    a[2]
a[0]   a[3]
*/
__pure inline void sort_by_xy_wall_2(c_dp4_t* a)
{
	for (int i = 1; i < 4; i++)
	{
		for (int j = i; j > 0 && a[j - 1].x > a[j].x; j--)
		{
			double t = a[j].x;
			a[j].x = a[j - 1].x;
			a[j - 1].x = t;
			t = a[j].y;
			a[j].y = a[j - 1].y;
			a[j - 1].y = t;
			t = a[j].x_initial;
			a[j].x_initial = a[j - 1].x_initial;
			a[j - 1].x_initial = t;
			t = a[j].y_initial;
			a[j].y_initial = a[j - 1].y_initial;
			a[j - 1].y_initial = t;
		}
	}
}

__pure inline static bool try_get_slope_ratio(const c_dp_t& bv, const c_dp_t& uv, double& value)
{
	if (fabs(bv.x - uv.x) < 1e-12)
	{
		return false;
	}
	value = fabs((uv.y - bv.y) / (uv.x - bv.x)); // óãëîâîé êîýôôèöèåíò ïðÿìîé
	if (value < 1e-12)
	{
		return false;
	}
	return true;
}


__pure inline static c_dp_t get_intersection_point(const c_dp4_t& alpha, const c_dp4_t& beta, const c_dp4_t& gamma, const c_dp4_t& theta)
{
	double a1 = gamma.y - alpha.y;
	double b1 = alpha.x - gamma.x; //double b1 = -(gamma.x - alpha.x);
	double c1 = a1 * alpha.x + b1 * alpha.y;
	double a2 = theta.y - beta.y;
	double b2 = beta.x - theta.x; //double b2 = -(theta.x - beta.x);
	double c2 = a2 * beta.x + b2 * beta.y;
	return c_dp_t((b1 * c2 - b2 * c1) / (b1 * a2 - b2 * a1), (a1 * c2 - a2 * c1) / (-b1 * a2 + b2 * a1));
}

__pure inline static double sign(const c_dp4_t& p1, const c_dp4_t p2, const c_dp4_t p3)
{
	return (p1.x - p3.x) * (p2.y - p3.y) - (p2.x - p3.x) * (p1.y - p3.y);
}

__pure inline static bool is_points_belong_to_one_line(const c_dp4_t& p1, const c_dp4_t p2, const c_dp4_t p3)
{
	return sign(p1, p2, p3) == FLT_MIN ;
}

__host__ __pure inline static double analytical_solution(double t, double x, double y)
{
	return 1.1 + sin(t * x * y);
}

__pure inline static double func_u(double b, double x, double y)
{
	return b * y * (1 - y) * (M_PI_2 + atan(-x));
}

__pure inline static double func_u(double b, const c_dp_t& p)
{
	return func_u(b, p.x, p.y);
}

__pure inline static double func_v(double ub, double bb, double lb, double rb, double time, double x, double y)
{
	return atan(0.1 * (x - lb) * (x - rb) * (1 + time) * (y - ub) * (y - bb));
}

__pure inline static double func_v(double ub, double bb, double lb, double rb, double time, const c_dp_t& p)
{
	return func_v(ub, bb, lb, rb, time, p.x, p.y);
}

__pure inline static double func_f(double b, double time, double ub, double bb, double lb, double rb, double x, double y)
{
	double arg_v = 0.1 * (x - lb) * (x - rb) * (1 + time) * (y - ub) * (y - bb);
	double rho = analytical_solution(time, x, y);
	double drho_dt = x * y * cos(time * x * y);
	double drho_dx = time * y * cos(time * x * y);
	double dtho_dy = time * x * cos(time * x * y);
	double u = func_u(b, x, y);
	double v = func_v(ub, bb, lb, rb, time, x, y);
	double du_dx = -b * y * (1 - y) / (1 + sqr(x));
	double dv_dx = 0.1 * (x - lb) * (x - rb) * (1 + time) * (y - bb + y - ub);
	dv_dx /= (1 + arg_v * arg_v);
	double res = drho_dt + rho * du_dx + u * drho_dx + rho * dv_dx + v * dtho_dy;
	return res;
}

__pure inline static double integrate_rectangle(double py, double qy, double gx, double hx, double a, double b)
{
	double t1 = __dmul_rn(hx-a, hx-a);
	t1 = t1 - __dmul_rn(gx-a, gx-a);
	double t3 = __dmul_rn(qy-b, qy-b);
	t3 = t3 - __dmul_rn(py-b, py-b);
	return __dmul_rn(__dmul_rn(0.25, t1), t3);
	//return 0.25 * (sqr(hx - a) - sqr(gx - a)) * (sqr(qy - b) - sqr(py - b));
}

__pure inline static double integrate_triangle(double py, double qy, double alpha, double beta, double a, double b)
{
	double x = __dadd_rn(__dadd_rn(__dmul_rn(a,qy), b), -beta);
	double xx = __dadd_rn(__dadd_rn(__dmul_rn(a,py), b), -beta);
	double x_cub = __dmul_rn(__dmul_rn(x,x),x);
	double xx_cub = __dmul_rn(__dmul_rn(xx,xx),xx);
	double x_quad = __dmul_rn(__dmul_rn(__dmul_rn(x,x),x),x);
	double xx_quad = __dmul_rn(__dmul_rn(__dmul_rn(xx,xx),xx),xx);
	double t1 = __dmul_rn(__dadd_rn(qy, -alpha), x_cub);
	double t2 = __dmul_rn(__dadd_rn(py, -alpha), xx_cub);
	double t3 = __dadd_rn(x_quad, -xx_quad);
	double t4 = __dmul_rn(6.0f,a);
	double t5 = __dmul_rn(24.0f,__dmul_rn(a,a));
	return ( (t1 - t2) / t4 - t3 / t5);
//	return (((qy - alpha) * cub(a * qy + b - beta) - (py - alpha) * cub(a * py + b - beta)) / (6 * a))
//		- (quad(a * qy + b - beta) - quad(a * py + b - beta)) / (24 * sqr(a));
}

__pure static double integrate_rectangle_one_cell(double* prev_dens, double py, double qy, double gx, double hx, const c_ip_t& sx, const c_ip_t& sy)
{
	double result, a, b;
	a = sx.y >= 0 && sy.y >= 0 ? OX_DEVICE[sx.y] : C_HX * sx.y;
	b = sx.y >= 0 && sy.y >= 0 ? OY_DEVICE[sy.y] : C_HY * sy.y; // ÝÒÎ ÏËÎÒÍÎÑÒÜ Ñ ÏÐÅÄÛÄÓÙÅÃÎ ÑËÎß ÄËß ÄÀÍÍÎÉ ß×ÅÉÊÈ
	result = integrate_rectangle(py, qy, gx, hx, a, b) * (sx.x >= 0 && sy.x >= 0 ? prev_dens[C_OX_LEN_1 * sy.x + sx.x] : analytical_solution(C_PREV_TIME, sx.x * C_HX, sy.x * C_HY));
	a = sx.x >= 0 && sy.y >= 0 ? OX_DEVICE[sx.x] : C_HX * sx.x;
	b = sx.x >= 0 && sy.y >= 0 ? OY_DEVICE[sy.y] : C_HY * sy.y;
	result -= integrate_rectangle(py, qy, gx, hx, a, b) * (sx.x >= 0 && sy.x >= 0 ? prev_dens[C_OX_LEN_1 * sy.x + sx.y] : analytical_solution(C_PREV_TIME, sx.y * C_HX, sy.x * C_HY));
	a = sx.y >= 0 && sy.x >= 0 ? OX_DEVICE[sx.y] : C_HX * sx.y;
	b = sx.y >= 0 && sy.x >= 0 ? OY_DEVICE[sy.x] : C_HY * sy.x;
	result -= integrate_rectangle(py, qy, gx, hx, a, b) * (sx.x >= 0 && sy.x >= 0 ? prev_dens[C_OX_LEN_1 * sy.y + sx.x] : analytical_solution(C_PREV_TIME, sx.x * C_HX, sy.y * C_HY));
	a = sx.x >= 0 && sy.x >= 0 ? OX_DEVICE[sx.x] : C_HX * sx.x;
	b = sx.x >= 0 && sy.x >= 0 ? OY_DEVICE[sy.x] : C_HY * sy.x;
	result += integrate_rectangle(py, qy, gx, hx, a, b) * (sx.x >= 0 && sy.x >= 0 ? prev_dens[C_OX_LEN_1 * sy.y + sx.y] : analytical_solution(C_PREV_TIME, sx.y * C_HX, sy.y * C_HY));
	
	return result * C_INVERTED_HX_HY;
}

__pure static double integrate_triangle_left_one_cell(double* prev_dens, const c_dp_t& bv, const c_dp_t& uv, double hx,
                                               const c_ip_t& sx, const c_ip_t& sy)
{
	double a_sl = (bv.x - uv.x) / (bv.y - uv.y); //   Coefficients of slant line: x = a_SL *y  +  b_SL.
	if (fabs(a_sl) <= FLT_MIN) return 0;
	double b_sl = uv.x - a_sl * uv.y;
	double result = 0, tmp, alpha, beta;
	alpha = sx.y >= 0 && sy.y >= 0 ? OY_DEVICE[sy.y] : C_HY * sy.y;
	beta = sx.y >= 0 && sy.y >= 0 ? OX_DEVICE[sx.y] : C_HX * sx.y;
	tmp = 0.25 * (sqr(uv.y - OY_DEVICE[sy.y]) - sqr(bv.y - OY_DEVICE[sy.y])) * sqr(hx - beta) - integrate_triangle(bv.y, uv.y, alpha, beta, a_sl, b_sl);
	
	result += tmp * (sx.x >= 0 && sx.y <= C_OX_LEN && sy.x >= 0 && sy.y <= C_OY_LEN ? prev_dens[C_OX_LEN_1 * sy.x + sx.x] : analytical_solution(C_PREV_TIME, sx.x * C_HX, sy.x * C_HY));
	
	beta = sx.x >= 0 && sy.y >= 0 ? OX_DEVICE[sx.x] : C_HX * sx.x;
	tmp = sqr(uv.y - OY_DEVICE[sy.y]) - sqr(bv.y - OY_DEVICE[sy.y]);
	tmp = -0.25 * tmp * sqr(hx - beta) + integrate_triangle(bv.y, uv.y, alpha, beta, a_sl, b_sl);

	result += tmp * (sx.x >= 0 && sx.y <= C_OX_LEN && sy.x >= 0 && sy.y <= C_OY_LEN ? prev_dens[C_OX_LEN_1 * sy.x + sx.y] : analytical_solution(C_PREV_TIME, sx.y * C_HX, sy.x * C_HY));
	
	alpha = sx.y >= 0 && sy.x >= 0 ? OY_DEVICE[sy.x] : C_HY * sy.x;
	beta = sx.y >= 0 && sy.x >= 0 ? OX_DEVICE[sx.y] : C_HX * sx.y;
	tmp = sqr(uv.y - OY_DEVICE[sy.x]) - sqr(bv.y - OY_DEVICE[sy.x]);
	tmp = -0.25 * tmp * sqr(hx - beta) + integrate_triangle(bv.y, uv.y, alpha, beta, a_sl, b_sl);
	result += tmp * (sx.x >= 0 && sx.y <= C_OX_LEN && sy.x >= 0 && sy.y <= C_OY_LEN ? prev_dens[C_OX_LEN_1 * sy.y + sx.x] : analytical_solution(C_PREV_TIME, sx.x * C_HX, sy.y * C_HY));
		
	alpha = sx.x >= 0 && sy.x >= 0 ? OY_DEVICE[sy.x] : C_HY * sy.x;
	beta = sx.x >= 0 && sy.x >= 0 ? OX_DEVICE[sx.x] : C_HX * sx.x;
	tmp = sqr(uv.y - OY_DEVICE[sy.x]) - sqr(bv.y - OY_DEVICE[sy.x]);
	tmp = 0.25 * tmp * sqr(hx - beta) - integrate_triangle(bv.y, uv.y, alpha, beta, a_sl, b_sl);
	result += tmp * (sx.x >= 0 && sx.y <= C_OX_LEN && sy.x >= 0 && sy.y <= C_OY_LEN ? prev_dens[C_OX_LEN_1 * sy.y + sx.y] : analytical_solution(C_PREV_TIME, sx.y * C_HX, sy.y * C_HY));	

	return result * C_INVERTED_HX_HY;
}

__pure static double integrate_right_slant_chanel(double* prev_dens, const c_dp_t& bv, const c_dp_t& uv, bool is_rect_truncated, const c_ip_t& sx, double b, const c_ip_t& sb, const c_ip_t& sy)
{
	if (fabs(uv.y - bv.y) <= FLT_MIN) return FLT_MIN ;
	double result = 0, gx = 0;
	double x = uv.x <= bv.x ? uv.x : bv.x;

	//   A. Under rectangle.
	result += -1 * integrate_triangle_left_one_cell(prev_dens, bv, uv, x, sx, sy);

	// case B: íåïîëíûé ïðÿìîóãîëüíèê    
	if (is_rect_truncated)
	{
		if (sx.x == sb.x) gx = b;
		if (sx.x > sb.x)
		{
			gx = sx.x >= 0 ? OX_DEVICE[sx.x] : C_HX * sx.x;
		}
		result += integrate_rectangle_one_cell(prev_dens, bv.y, uv.y, gx, x, sx, sy);
	}

	//   À òåïåðü ïðèáàâèì âñå ïðÿìîóãîëüíûå êóñêè, êîòîðûå ïîìåùàþòñÿ â ÿ÷åéêó
	c_ip_t ch_pos(sb.x, sb.x + 1);
	for (int j = sb.x; j < sx.x; j++)
	{
		if (j == sb.x) gx = b;
		else gx = ch_pos.x >= 0 ? OX_DEVICE[ch_pos.x] : C_HX * ch_pos.x;
		result += integrate_rectangle_one_cell(prev_dens, bv.y, uv.y, gx, ch_pos.x >= 0 ? OX_DEVICE[ch_pos.y] : C_HX * ch_pos.y, ch_pos, sy);
		ch_pos.x += 1;
		ch_pos.y = ch_pos.x + 1;
	}
	return result;
}

// èñïîëüçóåòñÿ äëÿ upper left è äëÿ bottom left òðåóãîëüíèêà
// ò.å. ñëó÷àé
// UPPERLEFTTR
//
//                  CENTRE
//
// BOTTOMLEFTTR

__pure static double integrate_left_slant_chanel(double* prev_dens, const c_dp_t& bv, const c_dp_t& uv,
                                          bool is_rect_trunc, const c_ip_t& sx, const c_ip_t& sy,
                                          double b, const c_ip_t& sb)
{
	if (fabs(uv.y - bv.y) <= FLT_MIN) return FLT_MIN;
	double result = 0, hx = 0; //   -  Left and right boundary for each integration.   
	double x = uv.x <= bv.x ? bv.x : uv.x;

	// case A: triangle
	result += integrate_triangle_left_one_cell(prev_dens, bv, uv, x, sx, sy);

	// case B: íå ïîëíûé ïðÿìîóãîëüíèê
	if (is_rect_trunc)
	{ // ýòî çíà÷èò, ÷òî ïðÿìîóãîëüíèê çàíèìàåò íå âñþ ÿ÷åéêó  
		hx = sx.x == sb.x ? b : (sx.y >= 0 ? OX_DEVICE[sx.y] : C_HX * sx.y);
		result += integrate_rectangle_one_cell(prev_dens, bv.y, uv.y, x, hx, sx, sy);
	}

	//   À òåïåðü ïðèáàâèì âñå ïðÿìîóãîëüíûå êóñêè, êîòîðûå ïîìåùàþòñÿ â ÿ÷åéêó
	c_ip_t ch_pos(sx.x + 1, sx.x + 2); //   - êîîðäèíàòû êàíàëà
	for (int j = sx.x + 1; j < sb.x + 1; j++)
	{
		hx = ch_pos.y <= 0 ? C_HX * ch_pos.y : hx = OX_DEVICE[ch_pos.y];
		if (j == sb.x) hx = b;
		result += integrate_rectangle_one_cell(prev_dens, bv.y, uv.y, ch_pos.y <= 0 ? C_HX * ch_pos.x : OX_DEVICE[ch_pos.x], hx, ch_pos, sy);
		ch_pos.x += 1;
		ch_pos.y = ch_pos.x + 1;
	}
	return result;
}

// îïðåäåëèì öåëî÷èñëåííûå èíäåêñû êâàäðàòîâ â êîòîðûõ ëåæàò âåðõíÿÿ è íèæíÿÿ òî÷êè òðåóãîëüíèêà
// sx = (x,y) êîîðäèíàòû êâàäðàòà â êîòîðîé ëåæèò íèæíÿÿ òî÷êà
// sy = (x,y) êîîðäèíàòû êâàäðàòà â êîòîðîé ëåæèò âåðõíÿÿ òî÷êà
// â ñëó÷àå óñïåøíîé ïðîâåðêè, k = áóäåò  óãëîâîé êîýôèöèåíò ïðÿìîé

__pure static double integrate_right_triangle_bottom_left(double* prev_dens, const c_dp_t& bv, const c_dp_t& uv)
{
	double k = 0;
	if (!try_get_slope_ratio(bv, uv, k)) return k;

	//   -  Index of current square by Ox and Oy axes. 
	c_ip_t sx, sy;
	sx.x = static_cast<int>((bv.x - FLT_MIN) / C_HX);
	if (bv.x - FLT_MIN <= 0) sx.x -= 1;
	sx.y = sx.x + 1;
	sy.x = static_cast<int>((bv.y + FLT_MIN) / C_HY);
	if (bv.y + FLT_MIN <= 0) sy.x -= 1;
	sy.y = sy.x + 1;

	c_ip_t ib(sx.x, sx.x + 1);
	double result = 0;
	int curr_i = 0, next_i;
	c_dp_t curr = bv, next;
	while (true)
	{
		//TODO: sx.x è sx.y äîëæíû áûòü ïîëîæèòåëüíûìè âñåãäà? Êàæåòñÿ äëÿ sx.x ýòî âñåãäà âåðíî...
		double slope = sx.y >= 0 ? OY_DEVICE[sy.y] - curr.y : fabs(C_HY * sy.y - curr.y);
		slope /= sx.x >= 0 ? curr.x - OX_DEVICE[sx.x] : fabs(curr.x - C_HX * sx.x);
		if (slope <= k)
		{
			next_i = 1;
			next.y = sy.y >= 0 ? OY_DEVICE[sy.y] : C_HY * sy.y;
			next.x = curr.x - (next.y - curr.y) / k;
		}
		else
		{
			next_i = 2;
			next.x = sx.x >= 0 ? OX_DEVICE[sx.x] : C_HX * sx.x;
			next.y = curr.y - k * (next.x - curr.x);
		}
		if (next.x - uv.x < FLT_MIN)
		{
			// ñþäà ïîïàäàåì è â ñëó÷àå êîãäà òðåóãîëüíèê ïîëíîñòüþ â îäíîé ÿ÷åéêå ëåæèò
			// è â ñëó÷àå êîãäà ïðîøëèñü ïî âñåì òî÷êàì...
			result += integrate_left_slant_chanel(prev_dens, curr, uv, (uv.x <= curr.x ? curr_i : 0) == 1, sx, sy, bv.x, ib);
			break;
		}
		result += integrate_left_slant_chanel(prev_dens, curr, next, (next.x <= curr.x ? curr_i : next_i) == 1, sx, sy, bv.x, ib);
		switch (next_i)
		{
		case 1:
			sy += 1;
			break;
		case 2:
			sx -= 1;
			break;
		}
		curr_i = next_i;
		curr = next;
	}
	return result;
}

__pure static double integrate_right_triangle_bottom_right(double* prev_dens, const c_dp_t& bv, const c_dp_t& uv)
{
	double k = 0;
	if (!try_get_slope_ratio(bv, uv, k)) return k;

	c_ip_t sx, sy;
	sx.x = static_cast<int>((bv.x + FLT_MIN) / C_HX);
	if (bv.x + FLT_MIN <= 0) sx.x -= 1;
	sx.y = sx.x + 1;
	sy.x = static_cast<int>((bv.y + FLT_MIN) / C_HY);
	if (bv.y + FLT_MIN <= 0) sy.x -= 1;
	sy.y = sy.x + 1;

	c_ip_t ib(sx.x, sx.x + 1);
	double result = 0;
	int curr_i = 0, next_i;
	c_dp_t curr = bv, next;
	while (true)
	{
		double slope = sy.y >= 0 ? fabs(OY_DEVICE[sy.y] - curr.y) : fabs(C_HY * sy.y - curr.y);
		slope /= sx.y >= 0 ? fabs(OX_DEVICE[sx.y] - curr.x) : fabs(C_HX * sx.y - curr.x);
		if (slope <= k)
		{
			next_i = 1;
			next.y = sy.y >= 0 ? OY_DEVICE[sy.y] : C_HY * sy.y;
			next.x = bv.x + (next.y - bv.y) / k;
		}
		else
		{
			next_i = 2;
			next.x = sx.y >= 0 ? OX_DEVICE[sx.y] : C_HX * sx.y;
			next.y = bv.y + k * (next.x - bv.x);
		}
		if (next.x - uv.x > FLT_MIN)
		{
			result += integrate_right_slant_chanel(prev_dens, curr, uv, (uv.x <= curr.x ? 0 : curr_i) == 1, sx, bv.x, ib, sy);
			break;
		}
		result += integrate_right_slant_chanel(prev_dens, curr, next, (next.x <= curr.x ? next_i : curr_i) == 1, sx, bv.x, ib, sy);
		switch (next_i)
		{
		case 1:
			sy += 1;
			break;
		case 2:
			sx += 1;
			break;
		}
		curr_i = next_i;
		curr = next;
	}
	return result;
}

__pure static double integrate_right_triangle_upper_left(double* prev_dens, const c_dp_t& bv, const c_dp_t& uv)
{
	double k = 0;
	if (!try_get_slope_ratio(bv, uv, k)) return k;

	c_ip_t sx, sy, ib;
	sx.x = static_cast<int>((bv.x + FLT_MIN) / C_HX); //   -  If bv.x is in grid edge I want it will be in the right side.
	if (bv.x + FLT_MIN <= 0) sx.x -= 1;
	sx.y = sx.x + 1;
	sy.x = static_cast<int>((bv.y + FLT_MIN) / C_HY); //   -  If bv.y is in grid edge I want it will be in the upper square.
	if (bv.y + FLT_MIN <= 0) sy.x -= 1;
	sy.y = sy.x + 1;
	ib.x = static_cast<int>((uv.x - FLT_MIN) / C_HY); //   -  If uv.x is in grid edge I want it will be in the left side.
	if (uv.x - FLT_MIN <= 0) ib.x -= 1;
	ib.y = ib.x + 1;

	double result = 0;
	int curr_i = 0, next_i;
	c_dp_t curr = bv, next;
	while (true)
	{
		double slope = sy.y >= 0 ? OY_DEVICE[sy.y] - curr.y : fabs(C_HY * sy.y - curr.y);
		slope /= sx.y >= 0 ? OX_DEVICE[sx.y] - curr.x : fabs(C_HX * sx.y - curr.x);
		if (slope <= k)
		{
			next_i = 1;
			next.y = sy.y >= 0 ? OY_DEVICE[sy.y] : C_HY * sy.y;
			next.x = bv.x + (next.y - bv.y) / k;
		}
		else
		{
			next_i = 2;
			next.x = sx.y >= 0 ? OX_DEVICE[sx.y] : C_HX * sx.y;
			next.y = bv.y + k * (next.x - bv.x);
		}
		if (next.x - uv.x > FLT_MIN) // åñëè ñëåäóþùàÿ òî÷êà óæå ïðàâåå, ÷åì íàøà ãðàíè÷íàÿ òî÷êà, òî ìû îáðàáîòàëè êàíàë
		{
			result += integrate_left_slant_chanel(prev_dens, curr, uv, (uv.x <= curr.x ? curr_i : 0) == 1, sx, sy, uv.x, ib);
			break;
		}
		result += integrate_left_slant_chanel(prev_dens, curr, next, (next.x <= curr.x ? curr_i : next_i) == 1, sx, sy, uv.x, ib);

		switch (next_i)
		{
		case 1:
			sy += 1;
			break;
		case 2:
			sx += 1;
			break;
		}
		curr_i = next_i;
		curr = next;
	}
	return result;
}

__pure static double integrate_right_triangle_upper_right(double* prev_dens, const c_dp_t& bv, const c_dp_t& uv)
{
	double k = 0;
	if (!try_get_slope_ratio(bv, uv, k)) return k;

	c_ip_t sx, sy, ib;
	sx.x = static_cast<int>((bv.x - FLT_MIN) / C_HX); //   -  If bv.x is in grid edge I want it will be between in the left side.
	if (bv.x - FLT_MIN <= 0) sx.x -= 1;
	sx.y = sx.x + 1;
	sy.x = static_cast<int>((bv.y + FLT_MIN) / C_HY); //   -  If bv.y is in grid edge I want it will be in the upper side.
	if (bv.y + FLT_MIN <= 0) sy.x -= 1;
	sy.y = sy.x + 1;
	ib.x = static_cast<int>((uv.x + FLT_MIN) / C_HX);
	if (uv.x + FLT_MIN <= 0) ib.x -= 1;
	ib.y = ib.x + 1;

	double result = 0;
	int curr_i = 0, next_i;
	c_dp_t curr = bv, next;
	while (true)
	{
		double slope = sy.y >= 0 ? fabs(OY_DEVICE[sy.y] - curr.y) : fabs(C_HY * sy.y - curr.y);
		slope /= sx.x >= 0 ? fabs(curr.x - OX_DEVICE[sx.x]) : fabs(curr.x - C_HX * sx.x);
		if (slope <= k)
		{
			next_i = 1;
			next.y = sy.y >= 0 ? OY_DEVICE[sy.y] : C_HY * sy.y;
			next.x = bv.x - (next.y - bv.y) / k;
		}
		else
		{
			next_i = 2;
			next.x = sx.x >= 0 ? OX_DEVICE[sx.x] : C_HX * sx.x;
			next.y = bv.y - k * (next.x - bv.x);
		}
		if (next.x - uv.x < FLT_MIN)
		{
			result += integrate_right_slant_chanel(prev_dens, curr, uv, (uv.x <= curr.x ? 0 : curr_i) == 1, sx, uv.x, ib, sy);
			break;
		}
		result += integrate_right_slant_chanel(prev_dens, curr, next, (next.x <= curr.x ? next_i : curr_i) == 1, sx, uv.x, ib, sy);
		switch (next_i)
		{
		case 1:
			sy += 1;
			break;
		case 2:
			sx -= 1;
			break;
		}
		curr_i = next_i;
		curr = next;
	}
	return result;
}

__pure static double integrate_bottom_triangle(double* prev_dens, const c_dp_t& l, const c_dp_t& m, const c_dp_t& r)
{
	double result = 0;
	if (m.x == l.x)
	{
		result = integrate_right_triangle_bottom_right(prev_dens, m, r);
	}
	else if (m.x == r.x)
	{
		result = integrate_right_triangle_bottom_left(prev_dens, m, l);
	}
	else if (m.x < l.x)
	{
		result = integrate_right_triangle_bottom_right(prev_dens, m, r) - integrate_right_triangle_bottom_right(prev_dens, m, l);
	}
	else if (m.x > l.x && m.x < r.x)
	{
		result = integrate_right_triangle_bottom_left(prev_dens, m, l) + integrate_right_triangle_bottom_right(prev_dens, m, r);
	}
	else if (m.x > r.x)
	{
		result = integrate_right_triangle_bottom_left(prev_dens, m, l) - integrate_right_triangle_bottom_left(prev_dens, m, r);
	}
	return result;
}

__pure static double integrate_upper_triangle(double* prev_dens, const c_dp_t& l, const c_dp_t& m, const c_dp_t& r)
{
	double result = 0;
	if (m.x == l.x)
	{
		result = integrate_right_triangle_upper_right(prev_dens, r, m);
	}
	else if (m.x == r.x)
	{
		result = integrate_right_triangle_upper_left(prev_dens, l, m);
	}
	else if (m.x < l.x)
	{
		result = integrate_right_triangle_upper_right(prev_dens, r, m) - integrate_right_triangle_upper_right(prev_dens, l, m);
	}
	else if (m.x > l.x && m.x < r.x)
	{
		result = integrate_right_triangle_upper_left(prev_dens, l, m) + integrate_right_triangle_upper_right(prev_dens, r, m);
	}
	else if (m.x > r.x)
	{
		result = integrate_right_triangle_upper_left(prev_dens, l, m) - integrate_right_triangle_upper_left(prev_dens, r, m);
	}
	return result;
}

// x,y,z
__pure static double integrate_uniform_triangle(double* prev_dens, const c_dp_t& x, const c_dp_t& y, const c_dp_t& z)
{
	// òî÷êè äîëæíû èäòè â ïîðÿäêå âîçðàñòàíèÿ y êîîðäèíàòû, ÷òîáû ïðàâèëüíî îòðàáîòàëà ïðîöåäóðà èíòåãðèðîâàíèÿ		

	//   a * x  +  b * y  = c.
	double a = z.y - x.y;
	if (fabs(a) < FLT_MIN) return FLT_MIN;
	double b = x.x - z.x;
	double c = b * x.y + a * x.x;
	c_dp_t ip((c - b * y.y) / a, y.y);

	//   Âîçìîæíû 2 ñëó÷àÿ ðàñïîëîæåíèÿ òî÷êè ïåðåñå÷åíèÿ îòíîñèòåëüíî ñðåäíåé
	//   ñëåâà èëè ñïðàâà.
	//   åñäè ñðåäíÿÿ òî÷êà ñïðàâà îò òî÷êè ïåðåñå÷åíèÿ
	//   îáìåíÿåì ìåñòàìè  X êîîðäèíàòû, ÷òîáû èñïîëüçîâàòü îäèí êîä äëÿ ðàñ÷åòà
	c_dp_t t = y;
	if (t.x >= ip.x)
	{
		double tx = t.x;
		t.x = ip.x;
		ip.x = tx;
	}

	return integrate_upper_triangle(prev_dens, t, z, ip) + integrate_bottom_triangle(prev_dens, t, x, ip);
}

__pure static quad_type get_quadrangle_type(int i, int j,
                                     c_dp_t& a, c_dp_t& b, c_dp_t& c, c_dp_t& k, c_dp_t& m, c_dp_t& n, c_dp4_t* p)
{
	// TODO êàêîé ïîðÿäîê òóò âñå òàêè ïðåäïîëàãåòñÿ? ïðîòèâ ÷àñîâîé íà÷èíàÿ ñ âåðõíåé ëåâîé?	
	c_dp_t alpha((OX_DEVICE[i - 1] + OX_DEVICE[i]) * 0.5, (OY_DEVICE[j - 1] + OY_DEVICE[j]) * 0.5),
		beta((OX_DEVICE[i + 1] + OX_DEVICE[i]) * 0.5, (OY_DEVICE[j - 1] + OY_DEVICE[j]) * 0.5),
		gamma((OX_DEVICE[i + 1] + OX_DEVICE[i]) * 0.5, (OY_DEVICE[j + 1] + OY_DEVICE[j]) * 0.5),
		theta((OX_DEVICE[i - 1] + OX_DEVICE[i]) * 0.5, (OY_DEVICE[j + 1] + OY_DEVICE[j]) * 0.5);

	// get prev coordnates
	double u = func_u(C_B, alpha);
	double v = func_v(C_UB, C_BB, C_LB, C_RB, C_TIME, alpha);

	p[0].x = alpha.x - C_TAU * u;
	p[0].y = alpha.y - C_TAU * v; 
	v = func_v(C_UB, C_BB, C_LB, C_RB, C_TIME, beta);
	u = func_u(C_B, beta);
	p[1].x = beta.x - C_TAU * u;
	p[1].y = beta.y - C_TAU * v; 
	v = func_v(C_UB, C_BB, C_LB, C_RB, C_TIME, gamma);
	u = func_u(C_B, gamma);
	p[2].x = gamma.x - C_TAU * u;
	p[2].y = gamma.y - C_TAU * v; 
	v = func_v(C_UB, C_BB, C_LB, C_RB, C_TIME, theta);
	u = func_u(C_B, theta);
	p[3].x = theta.x - C_TAU * u;
	p[3].y = theta.y - C_TAU * v; 

	c_dp_t intersection = get_intersection_point(p[0], p[1], p[2], p[3]);
	if ((p[1].y - intersection.y) * (p[3].y - intersection.y) > 0) return pseudo; // ??
	if ((p[0].x - intersection.x) * (p[2].x - intersection.x) > 0) return pseudo; // ??	
	if (is_points_belong_to_one_line(p[0], p[1], p[3])) return pseudo;
	
	a = p[0];
	b = p[1];
	c = p[2];
	k = p[0];
	m = p[3];
	n = p[2];
	return normal;
}

__pure static double integrate(double* prev_dens, int i, int j)
{
	c_dp_t a1, b1, c1, a2, b2, c2;
        c_dp4_t* p = new c_dp4_t[6];
	quad_type type = get_quadrangle_type(i, j, a1, b1, c1, a2, b2, c2, p);
	delete[] p;

	switch (type)
	{	
	case normal:{		 
		double result = 0;
		double t = 0;
		sort_by_y_asc(a1, b1, c1);		
		t = integrate_uniform_triangle(prev_dens, a1, b1, c1);			
		result += t;
		sort_by_y_asc(a2, b2, c2);
		t = integrate_uniform_triangle(prev_dens, a2, b2, c2);
		result += t;
		return result;}		
	case concave:
	case convex:
	case pseudo:
		return -1;
	}
	return 0;
}

inline static double get_norm_of_error(double* density, double ts_count_mul_steps)
{
	double r = 0;
	for (int k = 1; k < OY_LEN; ++k)
		for (int j = 1; j < OX_LEN; ++j)
			r += fabs(analytical_solution(ts_count_mul_steps, OX[j], OY[k])
				- density[(OY_LEN + 1) * k + j]);
	return HX * HY * r;
}

inline static void init(double b, double lb, double rb, double bb, double ub,
                        double tau, int time_step_count, int ox_length, int oy_length)
{
	B = b;
	UB = ub;
	BB = bb;
	LB = lb;
	RB = rb;
	TAU = tau;
	TIME_STEP_CNT = time_step_count;
	XY_LEN = (ox_length + 1) * (oy_length + 1);
	OX_LEN = ox_length;
	OX_LEN_1 = ox_length + 1;
	OY_LEN = oy_length;
	OX = new double[OX_LEN_1];
	OY = new double[OY_LEN + 1];
	for (int i = 0; i <= OX_LEN; ++i) OX[i] = lb + i * (rb - lb) / OX_LEN;
	for (int i = 0; i <= OY_LEN; ++i) OY[i] = bb + i * (ub - bb) / OY_LEN;
	HX = OX[1] - OX[0];
	HY = OY[1] - OY[0];
	INVERTED_HX_HY = 1 / HX / HY;
}

inline static void clean()
{
	B = 0;
	UB = 0;
	BB = 0;
	LB = 0;
	RB = 0;
	TAU = 0;
	TIME = 0;
	OX_LEN = 0;
	OY_LEN = 0;
	OX_LEN_1 = 0;
	TIME_STEP_CNT = 0;
	XY_LEN = 0;
	HX = 0;
	HY = 0;
	INVERTED_HX_HY = 0;
	delete [] OX;
	delete [] OY;
}

__global__ void kernel(double* prev_result, double* result)
{	
	 for (int opt = blockIdx.x * blockDim.x + threadIdx.x; opt < C_XY_LEN; opt += blockDim.x * gridDim.x)
	 {		
	 	int i = opt % (C_OX_LEN + 1);
	 	int j = opt / (C_OY_LEN + 1);

	 	if (j == 0)  // bottom bound
	 	{
	 		 result[ opt ] = 1.1  +  sin( C_TIME * C_HX * j * C_BB );
	 	}
		else if (i == 0) // left bound
		{
			 result[ opt ] = 1.1  +  sin( C_TIME * C_HX * i * C_LB );
		}
		else if (j == C_OY_LEN) // upper bound
		{ 
			 result[ opt ] = 1.1  +  sin( C_TIME * C_HX * i * C_UB );
		}
		else if (i == C_OX_LEN) // right bound
		{ 
			 result[ opt ] = 1.1  +  sin(  C_TIME * C_HX * j * C_RB );
		}
		else if (i > 0 && j > 0 && j != C_OY_LEN && i != C_OX_LEN)
		{                   
			double t = integrate(prev_result, i, j);	
			result[ opt ] =  t * C_INVERTED_HX_HY;			
			result[ opt ] += C_TAU * func_f(C_B, C_TIME, C_UB, C_BB, C_LB, C_RB, OX_DEVICE[i], OY_DEVICE[j]);			
		}
	 }
}

__pure static double integrate_quad(double *prev_density, int i, int j)
{	
	c_dp_t left(OX_DEVICE[i-1], OY_DEVICE[j]);
	c_dp_t right(OX_DEVICE[i+1], OY_DEVICE[j]);
	c_dp_t up(OX_DEVICE[i], OY_DEVICE[j+1]);
	c_dp_t bottom(OX_DEVICE[i], OY_DEVICE[j-1]);
	c_dp_t center(OX_DEVICE[i], OY_DEVICE[j]);
	double u = func_u(C_B, center.x, center.y);
	double v = func_v(C_UB, C_BB, C_LB, C_RB, C_TIME, center.x, center.y);
	center.x = center.x - C_TAU * u;
	center.y = center.y - C_TAU * v;	
	
	// проверим случай вылета точки за левую границу
	if (center.x <= 0) // вылет за левую границу
	{
		c_dp_t center_tk(OX_DEVICE[i], OY_DEVICE[j]);
		// найдем точку (t, y) пересечения траектории и оси ординат
												
		double y_ = 0;
		double t_ = 0;
		if ( center_tk.x - center.x < FLT_MIN )
		{ 
			y_ = 0.5 * (center_tk.y + center_tk.y); 
		}
		else 
		{ 
			y_ = center_tk.y - center.x 
				* ((center_tk.y - center.y) / (center_tk.x - center.x)); 
		}		

		// найдем время t* в точке перечесения 
		// уравнение прямой для точки (y, t)
		// t - t1 / t2-t1 = y-y1/y2-y1
		// => t = t1 + (y-y1)*(t2-t1)/y2-y1
		// здесь center_tk = первая точка 
		// center = вторая
		// TAU = t2 - t1
		// TIME = время на K слое по времени
		t_ = C_TIME - C_TAU * ((y_-center_tk.y)/(center.y - center_tk.y));

		// посчитаем TAU* 
		double tau_ = C_TIME - t_;

		double u = func_u(C_B, left.x, left.y);
		double v = func_v(C_UB, C_BB, C_LB, C_RB, t_, left.x, left.y);
		left.x = left.x - tau_ * u;
		left.y = left.y - tau_ * v;
		u = func_u(C_B, right.x, right.y);
		v = func_v(C_UB, C_BB, C_LB, C_RB, t_, right.x, right.y);
		right.x = right.x - tau_ * u;
		right.y = right.y - tau_ * v;
		u = func_u(C_B, up.x, up.y);
		v = func_v(C_UB, C_BB, C_LB, C_RB, t_, up.x, up.y);
		up.x = up.x - tau_ * u;
		up.y = up.y - tau_ * v;
		u = func_u(C_B, bottom.x, bottom.y);
		v = func_v(C_UB, C_BB, C_LB, C_RB, t_, bottom.x, bottom.y);
		bottom.x = bottom.x - tau_ * u;
		bottom.y = bottom.y - tau_ * v;	
		u = func_u(C_B, center.x, center.y);
		v = func_v(C_UB, C_BB, C_LB, C_RB, t_, center.x, center.y);
		center.x = center_tk.x - tau_ * u;
		center.y = center_tk.y - tau_ * v;
		
		double w_x_ksi = 0.5 * ((right.x-center.x)/C_HX + (center.x - left.x)/C_HX);
                double w_y_ksi = 0.5 * ((right.y-center.y)/C_HX + (center.y - left.y)/C_HX);
	        double w_x_the = 0.5 * ((up.x-center.x)/C_HY + (center.x - bottom.x)/C_HY);    
                double w_y_the = 0.5 * ((up.y-center.y)/C_HY + (center.y - bottom.y)/C_HY);
	        double det = w_x_ksi*w_y_the - w_x_the*w_y_ksi;

		double rho =  analytical_solution(t_, 0, y_);
		return det * rho * C_INVERTED_HX_HY;
	}

	u = func_u(C_B, left.x, left.y);
	v = func_v(C_UB, C_BB, C_LB, C_RB, C_TIME, left.x, left.y);
	left.x = left.x - C_TAU * u;
	left.y = left.y - C_TAU * v;
	u = func_u(C_B, right.x, right.y);
	v = func_v(C_UB, C_BB, C_LB, C_RB, C_TIME, right.x, right.y);
	right.x = right.x - C_TAU * u;
	right.y = right.y - C_TAU * v;
	u = func_u(C_B, up.x, up.y);
	v = func_v(C_UB, C_BB, C_LB, C_RB, C_TIME, up.x, up.y);
	up.x = up.x - C_TAU * u;
	up.y = up.y - C_TAU * v;
	u = func_u(C_B, bottom.x, bottom.y);
	v = func_v(C_UB, C_BB, C_LB, C_RB, C_TIME, bottom.x, bottom.y);
	bottom.x = bottom.x - C_TAU * u;
	bottom.y = bottom.y - C_TAU * v;
	
	double w_x_ksi = 0.5*((right.x-center.x)/C_HX + (center.x - left.x)/C_HX);
        double w_y_ksi = 0.5*((right.y-center.y)/C_HX + (center.y - left.y)/C_HX);
        double w_x_the = 0.5*((up.x-center.x)/C_HY + (center.x - bottom.x)/C_HY);    
        double w_y_the = 0.5*((up.y-center.y)/C_HY + (center.y - bottom.y)/C_HY);
        double det = w_x_ksi*w_y_the - w_x_the *w_y_ksi;

	int x = floor(center.x / C_HX);	
	int y = floor(center.y / C_HY);	
	double rho = prev_density[y * C_OX_LEN_1 + x] * (center.x - OX_DEVICE[x + 1]) * (center.y - OY_DEVICE[y + 1]);
	rho -= prev_density[y * C_OX_LEN_1 + x + 1] * (center.x - OX_DEVICE[x]) * (center.y - OY_DEVICE[y + 1]);
	rho += prev_density[(y + 1) * C_OX_LEN_1 + x + 1] * (center.x - OX_DEVICE[x]) * (center.y - OY_DEVICE[y]);
	rho -= prev_density[(y + 1) * C_OX_LEN_1 + x] * (center.x - OX_DEVICE[x + 1]) * (center.y - OY_DEVICE[y]);    
	return det * rho * C_INVERTED_HX_HY;
}

__global__ void kernel_quad(double* prev_result, double* result)
{	
	 for (int opt = blockIdx.x * blockDim.x + threadIdx.x; opt < C_XY_LEN; opt += blockDim.x * gridDim.x)
	 {		
	 	int i = opt % (C_OX_LEN + 1);
	 	int j = opt / (C_OY_LEN + 1);
	 	
	 	if (j == 0)  // bottom bound
	 	{
	 		 result[ opt ] = 1.1  +  sin( C_TIME * C_HX * j * C_BB );
	 	}
		else if (i == 0) // left bound
		{
			 result[ opt ] = 1.1  +  sin( C_TIME * C_HX * i * C_LB );
		}
		else if (j == C_OY_LEN) // upper bound
		{ 
			 result[ opt ] = 1.1  +  sin( C_TIME * C_HX * i * C_UB );
		}
		else if (i == C_OX_LEN) // right bound
		{ 
			 result[ opt ] = 1.1  +  sin(  C_TIME * C_HX * j * C_RB );
		}
		else if (i > 0 && j > 0 && j != C_OY_LEN && i != C_OX_LEN)
		{                   			
			result[ opt ] =  integrate_quad(prev_result, i, j);						
			result[ opt ] += C_TAU * func_f(C_B, C_TIME, C_UB, C_BB, C_LB, C_RB, OX_DEVICE[i], OY_DEVICE[j]);					
		}
	 }
}

float solve_cuda(double* density)
{
	const int gridSize = 256;
	const int blockSize =  512; 
	double *result = NULL, *prev_result = NULL, *ox = NULL, *oy=NULL;
	int size = sizeof(double)*XY_LEN;
	double *prev_result_h = new double[XY_LEN];
	for (int j = 0; j < OY_LEN + 1; j++)
	{
		for (int i = 0; i < OX_LEN_1; i++)
		{
			prev_result_h[OX_LEN_1 * j + i] = analytical_solution(0, OX[i], OY[j]);
		}
	}

	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
        checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_TAU), &TAU, sizeof(double)));	
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_B), &B, sizeof(double)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_LB), &LB, sizeof(double)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_RB), &RB, sizeof(double)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_BB), &BB, sizeof(double)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_UB), &UB, sizeof(double)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_INVERTED_HX_HY), &INVERTED_HX_HY, sizeof(double)));	
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_HX), &HX, sizeof(double)));	
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_HY), &HY, sizeof(double)));	
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_OX_LEN_1), &OX_LEN_1, sizeof(int)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_XY_LEN), &XY_LEN, sizeof(int)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_OX_LEN), &OX_LEN, sizeof(int)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_OY_LEN), &OY_LEN, sizeof(int)));
	
	checkCuda(hipMalloc((void**)&(result), size) );
	checkCuda(hipMemset(result, 0, size) );
	checkCuda(hipMalloc((void**)&(prev_result), size) );
	checkCuda(hipMalloc((void**)&(ox), sizeof(ox)*(OX_LEN+1)));
	checkCuda(hipMalloc((void**)&(oy), sizeof(oy)*(OY_LEN+1)));
	checkCuda(hipMemcpy(ox, OX, sizeof(ox)*(OX_LEN + 1), hipMemcpyHostToDevice));	
	checkCuda(hipMemcpy(oy, OY, sizeof(oy)*(OY_LEN + 1), hipMemcpyHostToDevice));	
	checkCuda(hipMemcpy(prev_result, prev_result_h, size, hipMemcpyHostToDevice));	
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(OX_DEVICE), &ox, sizeof(ox)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(OY_DEVICE), &oy, sizeof(oy)));	

	hipEventRecord(start, 0);   

	TIME = 0;
	int tl = 0;
	int tempTl  = TIME_STEP_CNT -1;
        while(tl < tempTl)
	{
	    checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_PREV_TIME), &TIME, sizeof(double)));
            TIME = TAU * (tl+1);
	    checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_TIME), &TIME, sizeof(double)));	
	    kernel<<<gridSize, blockSize>>>(prev_result, result);

	    checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_PREV_TIME), &TIME, sizeof(double)));
            TIME = TAU * (tl+2);
	    checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_TIME), &TIME, sizeof(double)));	
	    kernel<<<gridSize, blockSize>>>(result, prev_result);		 		 
	    tl+=2;            
	}
	
	if (TIME_STEP_CNT%2==0)
		checkCuda(hipMemcpy(density, prev_result, size, hipMemcpyDeviceToHost));
	else
		checkCuda(hipMemcpy(density, result, size, hipMemcpyDeviceToHost));
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	time /= 1000; // to seconds
	//printf("Computation Time %f\n", time);
	hipFree(result);
	hipFree(prev_result);
	hipFree(ox);
	hipFree(oy);
	hipDeviceReset();
	delete[] prev_result_h;
	return time;
}

float solve_quad_cuda(double* density, float& time)
{
	const int gridSize = 256;
	const int blockSize =  512; 
	//const int gridSize = 1;
	//const int blockSize =  1;
	double *result = NULL, *prev_result = NULL, *ox = NULL, *oy=NULL;
	int size = sizeof(double)*XY_LEN;
	double *prev_result_h = new double[XY_LEN];
	for (int j = 0; j < OY_LEN + 1; j++)
	{
		for (int i = 0; i < OX_LEN_1; i++)
		{
			prev_result_h[OX_LEN_1 * j + i] = analytical_solution(0, OX[i], OY[j]);
		}
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
        checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_TAU), &TAU, sizeof(double)));	
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_B), &B, sizeof(double)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_LB), &LB, sizeof(double)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_RB), &RB, sizeof(double)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_BB), &BB, sizeof(double)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_UB), &UB, sizeof(double)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_INVERTED_HX_HY), &INVERTED_HX_HY, sizeof(double)));	
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_HX), &HX, sizeof(double)));	
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_HY), &HY, sizeof(double)));	
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_OX_LEN_1), &OX_LEN_1, sizeof(int)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_XY_LEN), &XY_LEN, sizeof(int)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_OX_LEN), &OX_LEN, sizeof(int)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_OY_LEN), &OY_LEN, sizeof(int)));
	
	checkCuda(hipMalloc((void**)&(result), size) );
	checkCuda(hipMemset(result, 0, size) );
	checkCuda(hipMalloc((void**)&(prev_result), size) );
	checkCuda(hipMalloc((void**)&(ox), sizeof(ox)*(OX_LEN+1)));
	checkCuda(hipMalloc((void**)&(oy), sizeof(oy)*(OY_LEN+1)));
	checkCuda(hipMemcpy(ox, OX, sizeof(ox)*(OX_LEN + 1), hipMemcpyHostToDevice));	
	checkCuda(hipMemcpy(oy, OY, sizeof(oy)*(OY_LEN + 1), hipMemcpyHostToDevice));	
	checkCuda(hipMemcpy(prev_result, prev_result_h, size, hipMemcpyHostToDevice));	
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(OX_DEVICE), &ox, sizeof(ox)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(OY_DEVICE), &oy, sizeof(oy)));	

	hipEventRecord(start, 0);   

	TIME = 0;
	int tl = 0;
	int tempTl  = TIME_STEP_CNT -1;

        while(tl < tempTl)
	{
	    checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_PREV_TIME), &TIME, sizeof(double)));
            TIME = TAU * (tl+1);
	    checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_TIME), &TIME, sizeof(double)));	
	    kernel_quad<<<gridSize, blockSize>>>(prev_result, result);

	    checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_PREV_TIME), &TIME, sizeof(double)));
            TIME = TAU * (tl+2);
	    checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_TIME), &TIME, sizeof(double)));	
	    kernel_quad<<<gridSize, blockSize>>>(result, prev_result);		 		 
	    tl+=2;            
	}
	
	if (TIME_STEP_CNT%2==0)
		checkCuda(hipMemcpy(density, prev_result, size, hipMemcpyDeviceToHost));
	else
		checkCuda(hipMemcpy(density, result, size, hipMemcpyDeviceToHost));
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	time /=  1000;
//	printf("Computation Time %f s\n", time/1000);
	hipFree(result);
	hipFree(prev_result);
	hipFree(ox);
	hipFree(oy);
	hipDeviceReset();
	delete[] prev_result_h;
	return time;
}

double* compute_density_cuda_internal(double b, double lb, double rb, double bb, double ub,
                        double tau, int time_step_count, int ox_length, int oy_length, double& norm, float& time)
{
#ifdef __NVCC__	
    init(b, lb, rb, bb, ub, tau, time_step_count, ox_length, oy_length);
	double* density = new double[XY_LEN];
//	print_params(B, LB, RB, BB, UB, TAU, TIME_STEP_CNT, OX_LEN, OY_LEN);
	time = solve_cuda(density);
	norm = get_norm_of_error(density, TIME_STEP_CNT * TAU);
	clean();
	return density;
#else
        return NULL;
#endif
}

double* compute_density_quad_cuda_internal(double b, double lb, double rb, double bb, double ub,
                        double tau, int time_step_count, int ox_length, int oy_length, double& norm, float& time)
{
#ifdef __NVCC__	
    init(b, lb, rb, bb, ub, tau, time_step_count, ox_length, oy_length);
	double* density = new double[XY_LEN];
//	print_params(B, LB, RB, BB, UB, TAU, TIME_STEP_CNT, OX_LEN, OY_LEN);
	time = solve_quad_cuda(density, time);
	norm = get_norm_of_error(density, TIME_STEP_CNT * TAU);
	clean();
	return density;
#else
        return NULL;
#endif
}

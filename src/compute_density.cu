#include "hip/hip_runtime.h"
#include "common.h"
#include "point.h"
#include "utils.h"
#include "compute_density_cuda.cuh"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hemi.h>
__constant__ double C_B;
__constant__ double C_LB;
__constant__ double C_RB;
__constant__ double C_UB;
__constant__ double C_BB;
__constant__ double C_INVERTED_HX_HY;
__constant__ int C_HX;
__constant__ int C_HY;
__constant__ int C_OY_LEN;
__constant__ int C_OX_LEN;
__constant__ int C_OX_LEN_1;
__constant__ int C_XY_LEN;
__constant__ double C_PREV_TIME; // tau * (tl - 1)
__constant__ double C_TIME;
__constant__ double C_TAU;
__device__ double *OX_DEVICE, *OY_DEVICE;

#define sqr(x) ((x)*(x))
#define cub(x) ((x)*(x)*(x))
#define quad(x) ((x)*(x)*(x)*(x))

#ifdef __NVCC__
#define __pure __device__
#endif

static double B; //-V707
static double UB; //-V707
static double BB; //-V707
static double LB; //-V707
static double RB; //-V707
static double TAU;
static int OX_LEN;
static int OX_LEN_1; // OX_LEN_1
static int OY_LEN;
static int XY_LEN;
static int TIME_STEP_CNT;
static double HX; //-V707
static double HY; //-V707
static double* OX; //-V707
static double* OY; //-V707
static double TIME;
static double INVERTED_HX_HY;

/*
__pure inline static void print_params_const(int index, int needed_index,
	double b,
	double lb,
	double rb,
	double bb,
	double ub,
	double tau,
	int tl,
	int tl_count,
	int ox_length,
	int oy_length) {
	if (index == needed_index) {
		printf("index = %d\n", index);
		printf("b = %f\n", b);
		printf("lbDom = %f\n", lb);
		printf("rbDom = %f\n", rb);
		printf("bbDom = %f\n", bb);
		printf("ubDom = %f\n", ub);
		printf("tau = %f\n", tau);
		printf("Time level count = %d\n", tl_count);
		printf("current time level = %d\n", tl);
		printf("ox length = %d\n", ox_length + 1);
		printf("oy length = %d\n", oy_length + 1);
	}
}*/

__pure inline static void sort_by_y_asc(c_dp_t& x, c_dp_t& y, c_dp_t& z)
{	
	double t;
	if (x.y < y.y)
	{
		if (z.y < x.y) 
		{
			//swap(x, z);
			double t = x.x;
			x.x = z.x;
			z.x = t;
			t = x.y;
			x.y = z.y;
			z.y = t;
		}
	}
	else
	{
		if (y.y < z.y) 
		{
			//swap(x, y);
			t = x.x;
			x.x = y.x;
			y.x = t;
			t = x.y;
			x.y = y.y;
			y.y = t;
		}
		else 
		{
			//swap(x, z);
			t = x.x;
			x.x = z.x;
			z.x = t;
			t = x.y;
			x.y = z.y;
			z.y = t;
		}
	}
	if (z.y < y.y) 
	{
		//swap(y, z);
		t = y.x;
		y.x = z.x;
		z.x = t;
		t = y.y;
		y.y = z.y;
		z.y = t;
	}
}

__pure inline void sort_by_y(c_dp_t* a)
{
	for (int i = 1; i < 4; i++)
	{
		for (int j = i; j > 0 && a[j - 1].y > a[j].y; j--)
		{
			double t = a[j].x;
			a[j].x = a[j - 1].x;
			a[j - 1].x = t;
			t = a[j].y;
			a[j].y = a[j - 1].y;
			a[j - 1].y = t;
		}
	}
}

__pure inline void sort_by_y_desc_3(c_dp4_t* a)
{
	for (int i = 2; i < 4; i++)
	{
		for (int j = i; j > 1 && a[j - 1].y < a[j].y; j--)
		{
			double t = a[j].x;
			a[j].x = a[j - 1].x;
			a[j - 1].x = t;
			t = a[j].y;
			a[j].y = a[j - 1].y;
			a[j - 1].y = t;
			t = a[j].x_initial;
			a[j].x_initial = a[j - 1].x_initial;
			a[j - 1].x_initial = t;
			t = a[j].y_initial;
			a[j].y_initial = a[j - 1].y_initial;
			a[j - 1].y_initial = t;
		}
	}
}

__pure inline void sort_by_x_asc(c_dp4_t* a)
{
	for (int i = 1; i < 4; i++)
	{
		for (int j = i; j > 0 && a[j - 1].x > a[j].x; j--)
		{
			double t = a[j].x;
			a[j].x = a[j - 1].x;
			a[j - 1].x = t;
			t = a[j].y;
			a[j].y = a[j - 1].y;
			a[j - 1].y = t;
			t = a[j].x_initial;
			a[j].x_initial = a[j - 1].x_initial;
			a[j - 1].x_initial = t;
			t = a[j].y_initial;
			a[j].y_initial = a[j - 1].y_initial;
			a[j - 1].y_initial = t;
		}
	}

	if (a[0].y > a[1].y)
	{
		double t = a[0].x;
		a[0].x = a[1].x;
		a[1].x = t;
		t = a[0].y;
		a[0].y = a[1].y;
		a[1].y = t;
		t = a[0].x_initial;
		a[0].x_initial = a[1].x_initial;
		a[1].x_initial = t;
		t = a[0].y_initial;
		a[0].y_initial = a[1].y_initial;
		a[1].y_initial = t;
	}
	if (a[2].y < a[3].y)
	{
		double t = a[2].x;
		a[2].x = a[3].x;
		a[3].x = t;
		t = a[2].y;
		a[2].y = a[3].y;
		a[3].y = t;

		t = a[2].x_initial;
		a[2].x_initial = a[3].x_initial;
		a[3].x_initial = t;
		t = a[2].x_initial;
		a[2].x_initial = a[3].x_initial;
		a[3].x_initial = t;
	}
}

// ���������� �������
/*

a[1]    a[2]
a[0]   a[3]
*/
__pure inline void sort_by_xy_wall_2(c_dp4_t* a)
{
	for (int i = 1; i < 4; i++)
	{
		for (int j = i; j > 0 && a[j - 1].x > a[j].x; j--)
		{
			double t = a[j].x;
			a[j].x = a[j - 1].x;
			a[j - 1].x = t;
			t = a[j].y;
			a[j].y = a[j - 1].y;
			a[j - 1].y = t;
			t = a[j].x_initial;
			a[j].x_initial = a[j - 1].x_initial;
			a[j - 1].x_initial = t;
			t = a[j].y_initial;
			a[j].y_initial = a[j - 1].y_initial;
			a[j - 1].y_initial = t;
		}
	}
}

__pure inline static bool try_get_slope_ratio(const c_dp_t& bv, const c_dp_t& uv, double& value)
{
	if (fabs(bv.x - uv.x) < 1e-12)
	{
		return false;
	}
	value = fabs((uv.y - bv.y) / (uv.x - bv.x)); // ������� ����������� ������
	if (value < 1e-12)
	{
		return false;
	}
	return true;
}


__pure inline static c_dp_t get_intersection_point(const c_dp4_t& alpha, const c_dp4_t& beta, const c_dp4_t& gamma, const c_dp4_t& theta)
{
	double a1 = gamma.y - alpha.y;
	double b1 = alpha.x - gamma.x; //double b1 = -(gamma.x - alpha.x);
	double c1 = a1 * alpha.x + b1 * alpha.y;
	double a2 = theta.y - beta.y;
	double b2 = beta.x - theta.x; //double b2 = -(theta.x - beta.x);
	double c2 = a2 * beta.x + b2 * beta.y;
	return c_dp_t((b1 * c2 - b2 * c1) / (b1 * a2 - b2 * a1), (a1 * c2 - a2 * c1) / (-b1 * a2 + b2 * a1));
}

// __pure inline static double sign(const c_dp_t& p1, const c_dp_t p2, const c_dp_t p3)
// {
// 	return (p1.x - p3.x) * (p2.y - p3.y) - (p2.x - p3.x) * (p1.y - p3.y);
// }

__pure inline static double sign(const c_dp4_t& p1, const c_dp4_t p2, const c_dp4_t p3)
{
	return (p1.x - p3.x) * (p2.y - p3.y) - (p2.x - p3.x) * (p1.y - p3.y);
}

__pure inline static bool is_points_belong_to_one_line(const c_dp4_t& p1, const c_dp4_t p2, const c_dp4_t p3)
{
	return sign(p1, p2, p3) == FLT_MIN ;
}

// __pure inline static bool is_point_in_triangle(c_dp_t pt, c_dp_t v1, c_dp_t v2, c_dp_t v3)
// {
// 	bool b1, b2, b3;
// 	b1 = sign(pt, v1, v2) < 0.0;
// 	b2 = sign(pt, v2, v3) < 0.0;
// 	b3 = sign(pt, v3, v1) < 0.0;
// 	return b1 == b2 && b2 == b3;
// }

__host__ __pure inline static double analytical_solution(double t, double x, double y)
{
	return 1.1 + sin(t * x * y);
}

__pure inline static double func_u(double b, double x, double y)
{
	return b * y * (1 - y) * (M_PI_2 + atan(-x));
}

__pure inline static double func_u(double b, const c_dp_t& p)
{
	return func_u(b, p.x, p.y);
}

__pure inline static double func_v(double ub, double bb, double lb, double rb, double time, double x, double y)
{
	return atan(0.1 * (x - lb) * (x - rb) * (1 + time) * (y - ub) * (y - bb));
}

__pure inline static double func_v(double ub, double bb, double lb, double rb, double time, const c_dp_t& p)
{
	return func_v(ub, bb, lb, rb, time, p.x, p.y);
}

__pure inline static double func_f(double b, double time, double ub, double bb, double lb, double rb, double x, double y)
{
	double arg_v = 0.1 * (x - lb) * (x - rb) * (1 + time) * (y - ub) * (y - bb);
	double rho = analytical_solution(time, x, y);
	double drho_dt = x * y * cos(time * x * y);
	double drho_dx = time * y * cos(time * x * y);
	double dtho_dy = time * x * cos(time * x * y);
	double u = func_u(b, x, y);
	double v = func_v(ub, bb, lb, rb, time, x, y);
	double du_dx = -b * y * (1 - y) / (1 + sqr(x));
	double dv_dx = 0.1 * (x - lb) * (x - rb) * (1 + time) * (y - bb + y - ub);
	dv_dx /= (1 + arg_v * arg_v);
	double res = drho_dt + rho * du_dx + u * drho_dx + rho * dv_dx + v * dtho_dy;
	// print_f_params()...
	return res;
}

__pure inline static double integrate_rectangle(double py, double qy, double gx, double hx, double a, double b)
{
	return 0.25 * (sqr(hx - a) - sqr(gx - a)) * (sqr(qy - b) - sqr(py - b));
}

__pure inline static double integrate_triangle(double py, double qy, double alpha, double beta, double a, double b)
{
	return (((qy - alpha) * cub(a * qy + b - beta) - (py - alpha) * cub(a * py + b - beta)) / (6 * a))
		- (quad(a * qy + b - beta) - quad(a * py + b - beta)) / (24 * sqr(a));
}
__device__ int flag;
__pure static double integrate_rectangle_one_cell(double* prev_dens, double py, double qy, double gx, double hx, const c_ip_t& sx, const c_ip_t& sy)
{
	double result, a, b;
	a = sx.y >= 0 && sy.y >= 0 ? OX_DEVICE[sx.y] : C_HX * sx.y;
	b = sx.y >= 0 && sy.y >= 0 ? OY_DEVICE[sy.y] : C_HY * sy.y; // ��� ��������� � ����������� ���� ��� ������ ������
	result = integrate_rectangle(py, qy, gx, hx, a, b) * (sx.x >= 0 && sy.x >= 0 ? prev_dens[C_OX_LEN_1 * sy.x + sx.x] : analytical_solution(C_PREV_TIME, sx.x * C_HX, sy.x * C_HY));
	a = sx.x >= 0 && sy.y >= 0 ? OX_DEVICE[sx.x] : C_HX * sx.x;
	b = sx.x >= 0 && sy.y >= 0 ? OY_DEVICE[sy.y] : C_HY * sy.y;
	result -= integrate_rectangle(py, qy, gx, hx, a, b) * (sx.x >= 0 && sy.x >= 0 ? prev_dens[C_OX_LEN_1 * sy.x + sx.y] : analytical_solution(C_PREV_TIME, sx.y * C_HX, sy.x * C_HY));
	a = sx.y >= 0 && sy.x >= 0 ? OX_DEVICE[sx.y] : C_HX * sx.y;
	b = sx.y >= 0 && sy.x >= 0 ? OY_DEVICE[sy.x] : C_HY * sy.x;
	result -= integrate_rectangle(py, qy, gx, hx, a, b) * (sx.x >= 0 && sy.x >= 0 ? prev_dens[C_OX_LEN_1 * sy.y + sx.x] : analytical_solution(C_PREV_TIME, sx.x * C_HX, sy.y * C_HY));
	a = sx.x >= 0 && sy.x >= 0 ? OX_DEVICE[sx.x] : C_HX * sx.x;
	b = sx.x >= 0 && sy.x >= 0 ? OY_DEVICE[sy.x] : C_HY * sy.x;
	result += integrate_rectangle(py, qy, gx, hx, a, b) * (sx.x >= 0 && sy.x >= 0 ? prev_dens[C_OX_LEN_1 * sy.y + sx.y] : analytical_solution(C_PREV_TIME, sx.y * C_HX, sy.y * C_HY));
	// if (flag == 1) 
	// {
	// 	printf("%s\n", "integrate_rectangle_one_cell");
	// 	printf("%lf\n", result);
	// }
	return result * C_INVERTED_HX_HY;
}

__pure static double integrate_triangle_left_one_cell(double* prev_dens, const c_dp_t& bv, const c_dp_t& uv, double hx,
                                               const c_ip_t& sx, const c_ip_t& sy)
{
	double a_sl = (bv.x - uv.x) / (bv.y - uv.y); //   Coefficients of slant line: x = a_SL *y  +  b_SL.
	if (fabs(a_sl) <= FLT_MIN) return 0;
	double b_sl = uv.x - a_sl * uv.y;
	double result = 0, tmp, alpha, beta;
	alpha = sx.y >= 0 && sy.y >= 0 ? OY_DEVICE[sy.y] : C_HY * sy.y;
	beta = sx.y >= 0 && sy.y >= 0 ? OX_DEVICE[sx.y] : C_HX * sx.y;
	tmp = 0.25 * (sqr(uv.y - OY_DEVICE[sy.y]) - sqr(bv.y - OY_DEVICE[sy.y])) * sqr(hx - beta) - integrate_triangle(bv.y, uv.y, alpha, beta, a_sl, b_sl);
	
	result += tmp * (sx.x >= 0 && sx.y <= C_OX_LEN && sy.x >= 0 && sy.y <= C_OY_LEN ? prev_dens[C_OX_LEN_1 * sy.x + sx.x] : analytical_solution(C_PREV_TIME, sx.x * C_HX, sy.x * C_HY));
	// if (flag == 1) 
	// {
	// 	printf("%s\n", "integrate_triangle_left_one_cell result 1");
	// 	printf("%lf\n", result);
	// }
	beta = sx.x >= 0 && sy.y >= 0 ? OX_DEVICE[sx.x] : C_HX * sx.x;
	tmp = sqr(uv.y - OY_DEVICE[sy.y]) - sqr(bv.y - OY_DEVICE[sy.y]);
	tmp = -0.25 * tmp * sqr(hx - beta) + integrate_triangle(bv.y, uv.y, alpha, beta, a_sl, b_sl);

	result += tmp * (sx.x >= 0 && sx.y <= C_OX_LEN && sy.x >= 0 && sy.y <= C_OY_LEN ? prev_dens[C_OX_LEN_1 * sy.x + sx.y] : analytical_solution(C_PREV_TIME, sx.y * C_HX, sy.x * C_HY));
	// if (flag == 1) 
	// {
	// 	printf("%s\n", "integrate_triangle_left_one_cell result 2");
	// 	printf("%lf\n", result);
	// }
	alpha = sx.y >= 0 && sy.x >= 0 ? OY_DEVICE[sy.x] : C_HY * sy.x;
	beta = sx.y >= 0 && sy.x >= 0 ? OX_DEVICE[sx.y] : C_HX * sx.y;
	tmp = sqr(uv.y - OY_DEVICE[sy.x]) - sqr(bv.y - OY_DEVICE[sy.x]);
	tmp = -0.25 * tmp * sqr(hx - beta) + integrate_triangle(bv.y, uv.y, alpha, beta, a_sl, b_sl);
	result += tmp * (sx.x >= 0 && sx.y <= C_OX_LEN && sy.x >= 0 && sy.y <= C_OY_LEN ? prev_dens[C_OX_LEN_1 * sy.y + sx.x] : analytical_solution(C_PREV_TIME, sx.x * C_HX, sy.y * C_HY));
		if (flag == 1) 
	{
		printf("%s\n", "integrate_triangle_left_one_cell result 3");
		printf("%lf\n", sy.y);
		printf("%lf\n", sx.x);
		printf("%lf\n", C_OX_LEN_1);
		printf("%lf\n", C_OX_LEN_1 * sy.y + sx.x);
		printf("%lf\n", prev_dens[C_OX_LEN_1 * sy.y + sx.x]);
		printf("%lf\n", result);
	}
	alpha = sx.x >= 0 && sy.x >= 0 ? OY_DEVICE[sy.x] : C_HY * sy.x;
	beta = sx.x >= 0 && sy.x >= 0 ? OX_DEVICE[sx.x] : C_HX * sx.x;
	tmp = sqr(uv.y - OY_DEVICE[sy.x]) - sqr(bv.y - OY_DEVICE[sy.x]);
	tmp = 0.25 * tmp * sqr(hx - beta) - integrate_triangle(bv.y, uv.y, alpha, beta, a_sl, b_sl);
	result += tmp * (sx.x >= 0 && sx.y <= C_OX_LEN && sy.x >= 0 && sy.y <= C_OY_LEN ? prev_dens[C_OX_LEN_1 * sy.y + sx.y] : analytical_solution(C_PREV_TIME, sx.y * C_HX, sy.y * C_HY));	
// if (flag == 1) 
// 	{
// 		printf("%s\n", "integrate_triangle_left_one_cell result 4");
// 		printf("%lf\n", tmp);
// 	}
	return result * C_INVERTED_HX_HY;
}

__pure static double integrate_right_slant_chanel(double* prev_dens, const c_dp_t& bv, const c_dp_t& uv, bool is_rect_truncated, const c_ip_t& sx, double b, const c_ip_t& sb, const c_ip_t& sy)
{
	if (fabs(uv.y - bv.y) <= FLT_MIN) return FLT_MIN ;
	double result = 0, gx = 0;
	double x = uv.x <= bv.x ? uv.x : bv.x;

	//   A. Under rectangle.
	result += -1 * integrate_triangle_left_one_cell(prev_dens, bv, uv, x, sx, sy);

	// case B: �������� �������������    
	if (is_rect_truncated)
	{
		if (sx.x == sb.x) gx = b;
		if (sx.x > sb.x)
		{
			gx = sx.x >= 0 ? OX_DEVICE[sx.x] : C_HX * sx.x;
		}
		result += integrate_rectangle_one_cell(prev_dens, bv.y, uv.y, gx, x, sx, sy);
	}

	//   � ������ �������� ��� ������������� �����, ������� ���������� � ������
	c_ip_t ch_pos(sb.x, sb.x + 1);
	for (int j = sb.x; j < sx.x; j++)
	{
		if (j == sb.x) gx = b;
		else gx = ch_pos.x >= 0 ? OX_DEVICE[ch_pos.x] : C_HX * ch_pos.x;
		result += integrate_rectangle_one_cell(prev_dens, bv.y, uv.y, gx, ch_pos.x >= 0 ? OX_DEVICE[ch_pos.y] : C_HX * ch_pos.y, ch_pos, sy);
		ch_pos.x += 1;
		ch_pos.y = ch_pos.x + 1;
	}
	return result;
}

// ������������ ��� upper left � ��� bottom left ������������
// �.�. ������
// UPPERLEFTTR
//
//                  CENTRE
//
// BOTTOMLEFTTR

__pure static double integrate_left_slant_chanel(double* prev_dens, const c_dp_t& bv, const c_dp_t& uv,
                                          bool is_rect_trunc, const c_ip_t& sx, const c_ip_t& sy,
                                          double b, const c_ip_t& sb)
{
	if (fabs(uv.y - bv.y) <= FLT_MIN) return FLT_MIN;
	double result = 0, hx = 0; //   -  Left and right boundary for each integration.   
	double x = uv.x <= bv.x ? bv.x : uv.x;

	// case A: triangle
	result += integrate_triangle_left_one_cell(prev_dens, bv, uv, x, sx, sy);

	// case B: �� ������ �������������
	if (is_rect_trunc)
	{ // ��� ������, ��� ������������� �������� �� ��� ������  
		hx = sx.x == sb.x ? b : (sx.y >= 0 ? OX_DEVICE[sx.y] : C_HX * sx.y);
		result += integrate_rectangle_one_cell(prev_dens, bv.y, uv.y, x, hx, sx, sy);
	}

	//   � ������ �������� ��� ������������� �����, ������� ���������� � ������
	c_ip_t ch_pos(sx.x + 1, sx.x + 2); //   - ���������� ������
	for (int j = sx.x + 1; j < sb.x + 1; j++)
	{
		hx = ch_pos.y <= 0 ? C_HX * ch_pos.y : hx = OX_DEVICE[ch_pos.y];
		if (j == sb.x) hx = b;
		result += integrate_rectangle_one_cell(prev_dens, bv.y, uv.y, ch_pos.y <= 0 ? C_HX * ch_pos.x : OX_DEVICE[ch_pos.x], hx, ch_pos, sy);
		ch_pos.x += 1;
		ch_pos.y = ch_pos.x + 1;
	}
	return result;
}

// ��������� ������������� ������� ��������� � ������� ����� ������� � ������ ����� ������������
// sx = (x,y) ���������� �������� � ������� ����� ������ �����
// sy = (x,y) ���������� �������� � ������� ����� ������� �����
// � ������ �������� ��������, k = �����  ������� ���������� ������

__pure static double integrate_right_triangle_bottom_left(double* prev_dens, const c_dp_t& bv, const c_dp_t& uv)
{
	double k = 0;
	if (!try_get_slope_ratio(bv, uv, k)) return k;

	//   -  Index of current square by Ox and Oy axes. 
	c_ip_t sx, sy;
	sx.x = static_cast<int>((bv.x - FLT_MIN) / C_HX);
	if (bv.x - FLT_MIN <= 0) sx.x -= 1;
	sx.y = sx.x + 1;
	sy.x = static_cast<int>((bv.y + FLT_MIN) / C_HY);
	if (bv.y + FLT_MIN <= 0) sy.x -= 1;
	sy.y = sy.x + 1;

	c_ip_t ib(sx.x, sx.x + 1);
	double result = 0;
	int curr_i = 0, next_i;
	c_dp_t curr = bv, next;
	while (true)
	{
		//TODO: sx.x � sx.y ������ ���� �������������� ������? ������� ��� sx.x ��� ������ �����...
		double slope = sx.y >= 0 ? OY_DEVICE[sy.y] - curr.y : fabs(C_HY * sy.y - curr.y);
		slope /= sx.x >= 0 ? curr.x - OX_DEVICE[sx.x] : fabs(curr.x - C_HX * sx.x);
		if (slope <= k)
		{
			next_i = 1;
			next.y = sy.y >= 0 ? OY_DEVICE[sy.y] : C_HY * sy.y;
			next.x = curr.x - (next.y - curr.y) / k;
		}
		else
		{
			next_i = 2;
			next.x = sx.x >= 0 ? OX_DEVICE[sx.x] : C_HX * sx.x;
			next.y = curr.y - k * (next.x - curr.x);
		}
		if (next.x - uv.x < FLT_MIN)
		{
			// ���� �������� � � ������ ����� ����������� ��������� � ����� ������ �����
			// � � ������ ����� �������� �� ���� ������...
			result += integrate_left_slant_chanel(prev_dens, curr, uv, (uv.x <= curr.x ? curr_i : 0) == 1, sx, sy, bv.x, ib);
			break;
		}
		result += integrate_left_slant_chanel(prev_dens, curr, next, (next.x <= curr.x ? curr_i : next_i) == 1, sx, sy, bv.x, ib);
		switch (next_i)
		{
		case 1:
			sy += 1;
			break;
		case 2:
			sx -= 1;
			break;
		}
		curr_i = next_i;
		curr = next;
	}
	return result;
}

__pure static double integrate_right_triangle_bottom_right(double* prev_dens, const c_dp_t& bv, const c_dp_t& uv)
{
	double k = 0;
	if (!try_get_slope_ratio(bv, uv, k)) return k;

	c_ip_t sx, sy;
	sx.x = static_cast<int>((bv.x + FLT_MIN) / C_HX);
	if (bv.x + FLT_MIN <= 0) sx.x -= 1;
	sx.y = sx.x + 1;
	sy.x = static_cast<int>((bv.y + FLT_MIN) / C_HY);
	if (bv.y + FLT_MIN <= 0) sy.x -= 1;
	sy.y = sy.x + 1;

	c_ip_t ib(sx.x, sx.x + 1);
	double result = 0;
	int curr_i = 0, next_i;
	c_dp_t curr = bv, next;
	while (true)
	{
		double slope = sy.y >= 0 ? fabs(OY_DEVICE[sy.y] - curr.y) : fabs(C_HY * sy.y - curr.y);
		slope /= sx.y >= 0 ? fabs(OX_DEVICE[sx.y] - curr.x) : fabs(C_HX * sx.y - curr.x);
		if (slope <= k)
		{
			next_i = 1;
			next.y = sy.y >= 0 ? OY_DEVICE[sy.y] : C_HY * sy.y;
			next.x = bv.x + (next.y - bv.y) / k;
		}
		else
		{
			next_i = 2;
			next.x = sx.y >= 0 ? OX_DEVICE[sx.y] : C_HX * sx.y;
			next.y = bv.y + k * (next.x - bv.x);
		}
		if (next.x - uv.x > FLT_MIN)
		{
			result += integrate_right_slant_chanel(prev_dens, curr, uv, (uv.x <= curr.x ? 0 : curr_i) == 1, sx, bv.x, ib, sy);
			break;
		}
		result += integrate_right_slant_chanel(prev_dens, curr, next, (next.x <= curr.x ? next_i : curr_i) == 1, sx, bv.x, ib, sy);
		switch (next_i)
		{
		case 1:
			sy += 1;
			break;
		case 2:
			sx += 1;
			break;
		}
		curr_i = next_i;
		curr = next;
	}
	return result;
}

// __pure static double integrate_right_triangle_bottom_right_wall(const c_dp_t& bv, const c_dp_t& uv)
// {
// 	double k = 0;
// 	if (!try_get_slope_ratio(bv, uv, k)) return k;

// 	c_ip_t sx, sy;
// 	sx.x = static_cast<int>((bv.x + FLT_MIN) / C_HX);
// 	if (bv.x + FLT_MIN <= 0) sx.x -= 1;
// 	sx.y = sx.x + 1;
// 	sy.x = static_cast<int>((bv.y + FLT_MIN) / C_HY);
// 	if (bv.y + FLT_MIN <= 0) sy.x -= 1;
// 	sy.y = sy.x + 1;

// 	c_ip_t ib(sx.x, sx.x + 1);
// 	double result = 0;
// 	int curr_i = 0, next_i;
// 	c_dp_t curr = bv, next;
// 	while (true)
// 	{
// 		double slope = sy.y >= 0 ? fabs(OY_DEVICE[sy.y] - curr.y) : fabs(C_HY * sy.y - curr.y);
// 		slope /= sx.y >= 0 ? fabs(OX_DEVICE[sx.y] - curr.x) : fabs(C_HX * sx.y - curr.x);
// 		if (slope <= k)
// 		{
// 			next_i = 1;
// 			next.y = sy.y >= 0 ? OY_DEVICE[sy.y] : C_HY * sy.y;
// 			next.x = bv.x + (next.y - bv.y) / k;
// 		}
// 		else
// 		{
// 			next_i = 2;
// 			next.x = sx.y >= 0 ? OX_DEVICE[sx.y] : C_HX * sx.y;
// 			next.y = bv.y + k * (next.x - bv.x);
// 		}
// 		if (next.x - uv.x > FLT_MIN)
// 		{
// 			result += integrate_right_slant_chanel(curr, uv, (uv.x <= curr.x ? 0 : curr_i) == 1, sx, bv.x, ib, sy);
// 			break;
// 		}
// 		result += integrate_right_slant_chanel(curr, next, (next.x <= curr.x ? next_i : curr_i) == 1, sx, bv.x, ib, sy);
// 		switch (next_i)
// 		{
// 		case 1:
// 			sy += 1;
// 			break;
// 		case 2:
// 			sx += 1;
// 			break;
// 		}
// 		curr_i = next_i;
// 		curr = next;
// 	}
// 	return result;
// }

__pure static double integrate_right_triangle_upper_left(double* prev_dens, const c_dp_t& bv, const c_dp_t& uv)
{
	double k = 0;
	if (!try_get_slope_ratio(bv, uv, k)) return k;

	c_ip_t sx, sy, ib;
	sx.x = static_cast<int>((bv.x + FLT_MIN) / C_HX); //   -  If bv.x is in grid edge I want it will be in the right side.
	if (bv.x + FLT_MIN <= 0) sx.x -= 1;
	sx.y = sx.x + 1;
	sy.x = static_cast<int>((bv.y + FLT_MIN) / C_HY); //   -  If bv.y is in grid edge I want it will be in the upper square.
	if (bv.y + FLT_MIN <= 0) sy.x -= 1;
	sy.y = sy.x + 1;
	ib.x = static_cast<int>((uv.x - FLT_MIN) / C_HY); //   -  If uv.x is in grid edge I want it will be in the left side.
	if (uv.x - FLT_MIN <= 0) ib.x -= 1;
	ib.y = ib.x + 1;

	double result = 0;
	int curr_i = 0, next_i;
	c_dp_t curr = bv, next;
	while (true)
	{
		double slope = sy.y >= 0 ? OY_DEVICE[sy.y] - curr.y : fabs(C_HY * sy.y - curr.y);
		slope /= sx.y >= 0 ? OX_DEVICE[sx.y] - curr.x : fabs(C_HX * sx.y - curr.x);
		if (slope <= k)
		{
			next_i = 1;
			next.y = sy.y >= 0 ? OY_DEVICE[sy.y] : C_HY * sy.y;
			next.x = bv.x + (next.y - bv.y) / k;
		}
		else
		{
			next_i = 2;
			next.x = sx.y >= 0 ? OX_DEVICE[sx.y] : C_HX * sx.y;
			next.y = bv.y + k * (next.x - bv.x);
		}
		if (next.x - uv.x > FLT_MIN) // ���� ��������� ����� ��� ������, ��� ���� ��������� �����, �� �� ���������� �����
		{
			result += integrate_left_slant_chanel(prev_dens, curr, uv, (uv.x <= curr.x ? curr_i : 0) == 1, sx, sy, uv.x, ib);
			break;
		}
		result += integrate_left_slant_chanel(prev_dens, curr, next, (next.x <= curr.x ? curr_i : next_i) == 1, sx, sy, uv.x, ib);

		switch (next_i)
		{
		case 1:
			sy += 1;
			break;
		case 2:
			sx += 1;
			break;
		}
		curr_i = next_i;
		curr = next;
	}
	return result;
}

__pure static double integrate_right_triangle_upper_right(double* prev_dens, const c_dp_t& bv, const c_dp_t& uv)
{
	double k = 0;
	if (!try_get_slope_ratio(bv, uv, k)) return k;

	c_ip_t sx, sy, ib;
	sx.x = static_cast<int>((bv.x - FLT_MIN) / C_HX); //   -  If bv.x is in grid edge I want it will be between in the left side.
	if (bv.x - FLT_MIN <= 0) sx.x -= 1;
	sx.y = sx.x + 1;
	sy.x = static_cast<int>((bv.y + FLT_MIN) / C_HY); //   -  If bv.y is in grid edge I want it will be in the upper side.
	if (bv.y + FLT_MIN <= 0) sy.x -= 1;
	sy.y = sy.x + 1;
	ib.x = static_cast<int>((uv.x + FLT_MIN) / C_HX);
	if (uv.x + FLT_MIN <= 0) ib.x -= 1;
	ib.y = ib.x + 1;

	double result = 0;
	int curr_i = 0, next_i;
	c_dp_t curr = bv, next;
	while (true)
	{
		double slope = sy.y >= 0 ? fabs(OY_DEVICE[sy.y] - curr.y) : fabs(C_HY * sy.y - curr.y);
		slope /= sx.x >= 0 ? fabs(curr.x - OX_DEVICE[sx.x]) : fabs(curr.x - C_HX * sx.x);
		if (slope <= k)
		{
			next_i = 1;
			next.y = sy.y >= 0 ? OY_DEVICE[sy.y] : C_HY * sy.y;
			next.x = bv.x - (next.y - bv.y) / k;
		}
		else
		{
			next_i = 2;
			next.x = sx.x >= 0 ? OX_DEVICE[sx.x] : C_HX * sx.x;
			next.y = bv.y - k * (next.x - bv.x);
		}
		if (next.x - uv.x < FLT_MIN)
		{
			result += integrate_right_slant_chanel(prev_dens, curr, uv, (uv.x <= curr.x ? 0 : curr_i) == 1, sx, uv.x, ib, sy);
			break;
		}
		result += integrate_right_slant_chanel(prev_dens, curr, next, (next.x <= curr.x ? next_i : curr_i) == 1, sx, uv.x, ib, sy);
		switch (next_i)
		{
		case 1:
			sy += 1;
			break;
		case 2:
			sx -= 1;
			break;
		}
		curr_i = next_i;
		curr = next;
	}
	return result;
}

// __pure static double integrate_right_triangle_upper_left_wall(const c_dp_t& bv, const c_dp_t& uv)
// {
// 	double k = 0;
// 	if (!try_get_slope_ratio(bv, uv, k)) return k;

// 	c_ip_t sx, sy, ib;
// 	sx.x = static_cast<int>((bv.x + FLT_MIN) / C_HX); //   -  If bv.x is in grid edge I want it will be in the right side.
// 	if (bv.x + FLT_MIN <= 0) sx.x -= 1;
// 	sx.y = sx.x + 1;
// 	sy.x = static_cast<int>((bv.y + FLT_MIN) / C_HY); //   -  If bv.y is in grid edge I want it will be in the upper square.
// 	if (bv.y + FLT_MIN <= 0) sy.x -= 1;
// 	sy.y = sy.x + 1;
// 	ib.x = static_cast<int>((uv.x - FLT_MIN) / C_HY); //   -  If uv.x is in grid edge I want it will be in the left side.
// 	if (uv.x - FLT_MIN <= 0) ib.x -= 1;
// 	ib.y = ib.x + 1;

// 	double result = 0;
// 	int curr_i = 0, next_i;
// 	c_dp_t curr = bv, next;
// 	while (true)
// 	{
// 		double slope = sy.y >= 0 ? OY_DEVICE[sy.y] - curr.y : fabs(C_HY * sy.y - curr.y);
// 		slope /= sx.y >= 0 ? OX_DEVICE[sx.y] - curr.x : fabs(C_HX * sx.y - curr.x);
// 		if (slope <= k)
// 		{
// 			next_i = 1;
// 			next.y = sy.y >= 0 ? OY_DEVICE[sy.y] : C_HY * sy.y;
// 			next.x = bv.x + (next.y - bv.y) / k;
// 		}
// 		else
// 		{
// 			next_i = 2;
// 			next.x = sx.y >= 0 ? OX_DEVICE[sx.y] : C_HX * sx.y;
// 			next.y = bv.y + k * (next.x - bv.x);
// 		}
// 		if (next.x - uv.x > FLT_MIN) // ���� ��������� ����� ��� ������, ��� ���� ��������� �����, �� �� ���������� �����
// 		{
// 			result += integrate_left_slant_chanel(curr, uv, (uv.x <= curr.x ? curr_i : 0) == 1, sx, sy, uv.x, ib);
// 			break;
// 		}
// 		result += integrate_left_slant_chanel(curr, next, (next.x <= curr.x ? curr_i : next_i) == 1, sx, sy, uv.x, ib);

// 		switch (next_i)
// 		{
// 		case 1:
// 			sy += 1;
// 			break;
// 		case 2:
// 			sx += 1;
// 			break;
// 		}
// 		curr_i = next_i;
// 		curr = next;
// 	}
// 	return result;
// }

// __pure static double integrate_right_triangle_upper_right_wall(const c_dp_t& bv, const c_dp_t& uv)
// {
// 	double k = 0;
// 	if (!try_get_slope_ratio(bv, uv, k)) return k;

// 	c_ip_t sx, sy, ib;
// 	sx.x = static_cast<int>((bv.x - FLT_MIN) / C_HX); //   -  If bv.x is in grid edge I want it will be between in the left side.
// 	if (bv.x - FLT_MIN <= 0) sx.x -= 1;
// 	sx.y = sx.x + 1;
// 	sy.x = static_cast<int>((bv.y + FLT_MIN) / C_HY); //   -  If bv.y is in grid edge I want it will be in the upper side.
// 	if (bv.y + FLT_MIN <= 0) sy.x -= 1;
// 	sy.y = sy.x + 1;
// 	ib.x = static_cast<int>((uv.x + FLT_MIN) / C_HX);
// 	if (uv.x + FLT_MIN <= 0) ib.x -= 1;
// 	ib.y = ib.x + 1;

// 	double result = 0;
// 	int curr_i = 0, next_i;
// 	c_dp_t curr = bv, next;
// 	while (true)
// 	{
// 		double slope = sy.y >= 0 ? fabs(OY_DEVICE[sy.y] - curr.y) : fabs(C_HY * sy.y - curr.y);
// 		slope /= sx.x >= 0 ? fabs(curr.x - OX_DEVICE[sx.x]) : fabs(curr.x - C_HX * sx.x);
// 		if (slope <= k)
// 		{
// 			next_i = 1;
// 			next.y = sy.y >= 0 ? OY_DEVICE[sy.y] : C_HY * sy.y;
// 			next.x = bv.x - (next.y - bv.y) / k;
// 		}
// 		else
// 		{
// 			next_i = 2;
// 			next.x = sx.x >= 0 ? OX_DEVICE[sx.x] : C_HX * sx.x;
// 			next.y = bv.y - k * (next.x - bv.x);
// 		}
// 		if (next.x - uv.x < FLT_MIN)
// 		{
// 			result += integrate_right_slant_chanel(curr, uv, (uv.x <= curr.x ? 0 : curr_i) == 1, sx, uv.x, ib, sy);
// 			break;
// 		}
// 		result += integrate_right_slant_chanel(curr, next, (next.x <= curr.x ? next_i : curr_i) == 1, sx, uv.x, ib, sy);
// 		switch (next_i)
// 		{
// 		case 1:
// 			sy += 1;
// 			break;
// 		case 2:
// 			sx -= 1;
// 			break;
// 		}
// 		curr_i = next_i;
// 		curr = next;
// 	}
// 	return result;
// }

__pure static double integrate_bottom_triangle(double* prev_dens, const c_dp_t& l, const c_dp_t& m, const c_dp_t& r)
{
	double result = 0;
	if (m.x == l.x)
	{
		result = integrate_right_triangle_bottom_right(prev_dens, m, r);
	}
	else if (m.x == r.x)
	{
		result = integrate_right_triangle_bottom_left(prev_dens, m, l);
	}
	else if (m.x < l.x)
	{
		result = integrate_right_triangle_bottom_right(prev_dens, m, r) - integrate_right_triangle_bottom_right(prev_dens, m, l);
	}
	else if (m.x > l.x && m.x < r.x)
	{
		result = integrate_right_triangle_bottom_left(prev_dens, m, l) + integrate_right_triangle_bottom_right(prev_dens, m, r);
	}
	else if (m.x > r.x)
	{
		result = integrate_right_triangle_bottom_left(prev_dens, m, l) - integrate_right_triangle_bottom_left(prev_dens, m, r);
	}
	return result;
}

__pure static double integrate_upper_triangle(double* prev_dens, const c_dp_t& l, const c_dp_t& m, const c_dp_t& r)
{
	double result = 0;
	if (m.x == l.x)
	{
		result = integrate_right_triangle_upper_right(prev_dens, r, m);
	}
	else if (m.x == r.x)
	{
		result = integrate_right_triangle_upper_left(prev_dens, l, m);
	}
	else if (m.x < l.x)
	{
		result = integrate_right_triangle_upper_right(prev_dens, r, m) - integrate_right_triangle_upper_right(prev_dens, l, m);
	}
	else if (m.x > l.x && m.x < r.x)
	{
		result = integrate_right_triangle_upper_left(prev_dens, l, m) + integrate_right_triangle_upper_right(prev_dens, r, m);
	}
	else if (m.x > r.x)
	{
		result = integrate_right_triangle_upper_left(prev_dens, l, m) - integrate_right_triangle_upper_left(prev_dens, r, m);
	}
	return result;
}

// x,y,z
__pure static double integrate_uniform_triangle(double* prev_dens, const c_dp_t& x, const c_dp_t& y, const c_dp_t& z)
{
	// ����� ������ ���� � ������� ����������� y ����������, ����� ��������� ���������� ��������� ��������������		

	//   a * x  +  b * y  = c.
	double a = z.y - x.y;
	if (fabs(a) < FLT_MIN) return FLT_MIN;
	double b = x.x - z.x;
	double c = b * x.y + a * x.x;
	c_dp_t ip((c - b * y.y) / a, y.y);

	//   �������� 2 ������ ������������ ����� ����������� ������������ �������
	//   ����� ��� ������.
	//   ���� ������� ����� ������ �� ����� �����������
	//   �������� �������  X ����������, ����� ������������ ���� ��� ��� �������
	c_dp_t t = y;
	if (t.x >= ip.x)
	{
		double tx = t.x;
		t.x = ip.x;
		ip.x = tx;
	}

	return integrate_upper_triangle(prev_dens, t, z, ip) + integrate_bottom_triangle(prev_dens, t, x, ip);
}

// __pure static double integrate_uniform_triangle_wall(const c_dp_t& x, const c_dp_t& y,
//                                               const c_dp_t& z, quad_type type)
// {
// 	// ��� ����� YOt ��� ��������� ���������� �� �������
// 	// �� OX ����� ������������� �������� y
// 	// �� OY ����� ������������� �������� t
// 	// �.�. ����� ��������� YOt
// 	// ����� y ���������� ����� ������ ���� ��������	
// 	switch (type)
// 	{
// 	case wall_1_middle_at:
// 	case wall_1_middle_in:
// 	case wall_1_middle_out:		
// 			// !phd\2014\fem\ggb\wa1\4.ggb
// 			// ������ �
// 			if (x.x >= y.x)
// 			{
// 				double res = 0;
// 				double t = integrate_right_triangle_upper_left_wall(z, x);
// 				res += t;
// 				t = integrate_right_triangle_upper_left_wall(y, x);
// 				res += t;
// 				return res;
// 			}
// 			if (x.x < y.x && x.x > z.x) // ������ B
// 			{
// 				double res = 0;
// 				double t = integrate_right_triangle_upper_left_wall(z, x);
// 				res += t;
// 				t = integrate_right_triangle_upper_right_wall(y, x);
// 				res += t;
// 				return res;
// 			}
// 			if (x.x <= z.x) // ������ C
// 			{
// 				double res = 0;
// 				double t = integrate_right_triangle_upper_right_wall(y, x);
// 				res += t;
// 				t = integrate_right_triangle_upper_left_wall(z, x);
// 				res += t;
// 				return res;
// 			}
// 	case wall_2:
// 	{
// 		double t = 0;
// 		double res = 0;
// 		t = integrate_right_triangle_upper_right_wall(x, y);
// 		res += t;
// 		t = integrate_right_triangle_bottom_right_wall(y, z);
// 		res += t;
// 		return res;
// 	}
// 		break;

// 	default:
// 		return 0;
// 	}
// }

// __pure inline int get_wall_intersection_type_as_int(c_dp4_t* a)
// {
// 	int type = -1;
// 	bool is_four_point_on_the_wall = a[0].x <= 0 && a[1].x <= 0 && a[2].x <= 0 && a[3].x <= 0;
// 	bool is_three_point_on_the_wall = a[0].x <= 0 && a[1].x <= 0 && a[2].x <= 0 && a[3].x > 0;
// 	bool is_two_point_on_the_wall = a[0].x <= 0 && a[1].x <= 0 && a[2].x > 0 && a[3].x > 0;
// 	bool is_one_point_on_the_wall = a[0].x <= 0 && a[1].x > 0 && a[2].x > 0 && a[3].x > 0;
// 	if (is_four_point_on_the_wall)
// 	{
// 		type = 4;
// 	}
// 	else if (is_three_point_on_the_wall)
// 	{
// 		type = 3;
// 	}
// 	else if (is_two_point_on_the_wall)
// 	{
// 		type = 2;
// 	}
// 	else if (is_one_point_on_the_wall)
// 	{
// 		type = 1;
// 	}
// 	else
// 	{
// 		type = 0;
// 	}
// 	return type;
// }

// __pure inline static quad_type get_wall_intersection_type(c_dp4_t* a)
// {
// 	/*
// 	 ������� �������� ����� http://www.pm298.ru/reshenie/fha0327.php
// 	 a[0] - alpha
// 	 a[1] - beta
// 	 a[2] - gamma
// 	 a[3] - theta
// 	 a[4] - mu
// 	 a[5] - nu

// 	 */

// 	int type = get_wall_intersection_type_as_int(a);
// 	switch (type)
// 	{
// 	case 4:
// 		return wall_4;
// 	case 3:
// 		{
// 			sort_by_x_asc(a);
// 			sort_by_y_desc_3(a);
// 			// ���������� ����� ����������� OY � ������ a[0]:a[3]
// 			// ��� �� ���� fabs, ������ ��� a[3].x > a[0].x
// 			double y = a[3].x - a[0].x < FLT_MIN ? 0.5 * (a[0].y + a[3].y) : a[0].y - a[0].x * ((a[3].y - a[0].y) / (a[3].x - a[0].x));
// 			a[4] = c_dp4_t(0, y); // mu

// 			// ���������� ����� ����������� OY � ������ a[2]:a[3]
// 			// ��� �� ���� fabs, ������ ��� a[3].x > a[2].x
// 			y = a[3].x - a[2].x < FLT_MIN ? 0.5 * (a[3].y + a[2].y) : a[2].y - a[2].x * ((a[3].y - a[2].y) / (a[3].x - a[2].x));
// 			a[5] = c_dp4_t(0, y); // nu

// 			if ((a[0].x - a[2].x) * (a[1].y - a[2].y) - (a[1].x - a[2].x) * (a[0].y - a[2].y) < FLT_MIN)
// 				return wall_3_middle_at;
// 			if (a[0].x < a[1].x && a[1].x > a[2].x)
// 				return wall_3_middle_out;
// 			if (a[1].x < a[0].x && a[1].x < a[2].x)
// 				return wall_3_middle_in;
// 		}
// 	case 2:
// 		{
// 			sort_by_xy_wall_2(a);
// 			double y = 0;
// 			if (a[2].x - a[1].x < FLT_MIN)
// 			{
// 				y = (a[1].y + a[2].y) * 0.5;
// 			}
// 			else
// 			{
// 				y = a[1].y - a[1].x * ((a[2].y - a[1].y) / (a[2].x - a[1].x));
// 			}
// 			a[4] = c_dp4_t(0, y); // mu

// 			if (a[3].x - a[0].x < FLT_MIN)
// 			{
// 				y = (a[0].y + a[3].y) * 0.5;
// 			}
// 			else
// 			{
// 				y = a[0].y - a[0].x * ((a[3].y - a[0].y) / (a[3].x - a[0].x));
// 			}
// 			a[5] = c_dp4_t(0, y); // nu
// 			return wall_2;
// 		}

// 	case 1: 
// 		{
// 			sort_by_x_asc(a);
// 			sort_by_y_desc_3(a);

// 			// ��� ����� �� ������ ��� ��������� ���������� �� �������
// 			// �� OX ����� ������������� �������� y
// 			// �� OY ����� ������������� �������� t
// 			// ����� a[0] - �����, ������� ����� �� ������
// 			// ������� y ����������
// 			a[0].x = a[0].y_initial - a[0].x_initial * func_v(C_UB, C_BB, C_LB, C_RB, C_TIME, a[0].x_initial, a[0].y_initial) / func_u(C_B, a[0].x_initial, a[0].y_initial);
// 			a[0].y = C_TIME - a[0].x_initial*(1 / func_u(C_B, a[0].x_initial, a[0].y_initial)); // ����� ����� Mt = tk - Ax*tg(alpha); tg(alpha) = 1 / U(A)
			

// 			// ���������� ����� ����������� OY � ������ a[0]:a[1]
// 			// ��� �� ���� fabs, ������ ��� a[1].x > a[0].x
// 			double y = a[1].x - a[0].x < FLT_MIN ? 0.5 * (a[0].y + a[1].y) : (a[0].y - a[0].x * ((a[1].y - a[0].y) / (a[1].x - a[0].x)));
// 			a[4] = c_dp4_t(0, y); // mu

// 			// ���������� ����� ����������� OY � ������ a[0]:a[3]
// 			// ��� �� ���� fabs, ������ ��� a[3].x > a[0].x
// 			y = a[3].x - a[0].x < FLT_MIN ? 0.5 * (a[0].y + a[3].y) : (a[0].y - a[0].x * ((a[3].y - a[0].y) / (a[3].x - a[0].x)));
// 			a[5] = c_dp4_t(0, y); // nu

// 			if (is_points_belong_to_one_line(a[1], a[2], a[3]))
// 				return wall_1_middle_at;
// 			if (a[1].x < a[2].x && a[2].x > a[3].x)
// 				return wall_1_middle_out;
// 			if (a[2].x < a[1].x && a[2].x < a[3].x)
// 				return wall_1_middle_in;
// 			break;
// 		}
// 	default:
// 		return normal;
// 	}
// 	return normal;
// }

__pure static quad_type get_quadrangle_type(int i, int j,
                                     c_dp_t& a, c_dp_t& b, c_dp_t& c, c_dp_t& k, c_dp_t& m, c_dp_t& n, c_dp4_t* p)
{
	// TODO ����� ������� ��� ��� ���� �������������? ������ ������� ������� � ������� �����?	
	c_dp_t alpha((OX_DEVICE[i - 1] + OX_DEVICE[i]) * 0.5, (OY_DEVICE[j - 1] + OY_DEVICE[j]) * 0.5),
		beta((OX_DEVICE[i + 1] + OX_DEVICE[i]) * 0.5, (OY_DEVICE[j - 1] + OY_DEVICE[j]) * 0.5),
		gamma((OX_DEVICE[i + 1] + OX_DEVICE[i]) * 0.5, (OY_DEVICE[j + 1] + OY_DEVICE[j]) * 0.5),
		theta((OX_DEVICE[i - 1] + OX_DEVICE[i]) * 0.5, (OY_DEVICE[j + 1] + OY_DEVICE[j]) * 0.5);

	// get prev coordnates
	double u = func_u(C_B, alpha);
	double v = func_v(C_UB, C_BB, C_LB, C_RB, C_TIME, alpha);

	p[0].x = alpha.x - C_TAU * u;
	p[0].y = alpha.y - C_TAU * v;
	p[0].x_initial = alpha.x;
	p[0].y_initial = alpha.y;

	v = func_v(C_UB, C_BB, C_LB, C_RB, C_TIME, beta);
	u = func_u(C_B, beta);
	p[1].x = beta.x - C_TAU * u;
	p[1].y = beta.y - C_TAU * v;
	p[1].x_initial = beta.x;
	p[1].y_initial = beta.y;
	v = func_v(C_UB, C_BB, C_LB, C_RB, C_TIME, gamma);
	u = func_u(C_B, gamma);
	p[2].x = gamma.x - C_TAU * u;
	p[2].y = gamma.y - C_TAU * v;
	p[2].x_initial = gamma.x;
	p[2].y_initial = gamma.y;
	v = func_v(C_UB, C_BB, C_LB, C_RB, C_TIME, theta);
	u = func_u(C_B, theta);
	p[3].x = theta.x - C_TAU * u;
	p[3].y = theta.y - C_TAU * v;
	p[3].x_initial = theta.x;
	p[3].y_initial = theta.y;

	c_dp_t intersection = get_intersection_point(p[0], p[1], p[2], p[3]);
	if ((p[1].y - intersection.y) * (p[3].y - intersection.y) > 0) return pseudo; // ??
	if ((p[0].x - intersection.x) * (p[2].x - intersection.x) > 0) return pseudo; // ??	
	if (is_points_belong_to_one_line(p[0], p[1], p[3])) return pseudo;
	
	a = p[0];
	b = p[1];
	c = p[2];
	k = p[0];
	m = p[3];
	n = p[2];
return normal;
	//return get_wall_intersection_type(p);
}

// __pure static double integrate_wall_triangle(const c_dp_t wp, // wall point
//                                       double ly, // left y coordinate
//                                       double ry) // right y coordinate
// {
// 	return 0;
// }

// __pure static double integrate_wall_rectangle(const c_dp_t wp1, const c_dp_t wp2, const c_dp_t wp3, const c_dp_t wp4, double wp1y, double wp2y)
// {
// 	return 0;
// }

// __pure static double integrate_wall_rectangle(const c_dp_t wp1, const c_dp_t wp2, double wp1y, double wp2y)
// {
// 	return 0;
// }

// __pure static double integrate_wall_pentagon(const c_dp_t wp1, const c_dp_t wp2, const c_dp_t wp3, double y1, double y2)
// {
// 	return 0;
// }

// __pure static double integrate_pentagon(const c_dp_t x, const c_dp_t y, const c_dp_t z, double ly, double ry)
// {
// 	return 0;
// }

__pure static double integrate(double* prev_dens, int i, int j)
{
	c_dp_t a1, b1, c1, a2, b2, c2;
	c_dp4_t* p = new c_dp4_t[6];
	quad_type type = get_quadrangle_type(i, j, a1, b1, c1, a2, b2, c2, p);


	switch (type)
	{
	case wall_1_middle_in: // ������� ��� ���� � ��� �� ������
	case wall_1_middle_out:
	case wall_1_middle_at:
		{
//			//��� ���������� ������ 3 ������������
//			double result = 0;
//			double t = 0;			
//			c_dp_t v1 = c_dp_t(p[4].x, p[4].y);
//			c_dp_t v2 = c_dp_t(p[2].x, p[2].y);
//			c_dp_t v3 = c_dp_t(p[1].x, p[1].y);
//			sort_by_y_asc(v1, v2, v3);
//			t = integrate_uniform_triangle(v1, v2, v3);
//			result += t;
//										
//			v1 = c_dp_t(p[4].x, p[4].y);
//			v2 = c_dp_t(p[2].x, p[2].y);
//			v3 = c_dp_t(p[5].x, p[5].y);
//			sort_by_y_asc(v1, v2, v3);
//			t = integrate_uniform_triangle(v1, v2, v3); 
//			result += t;
//										
//			v1 = c_dp_t(p[3].x, p[3].y);
//			v2 = c_dp_t(p[2].x, p[2].y);
//			v3 = c_dp_t(p[5].x, p[5].y);
//			sort_by_y_asc(v1, v2, v3);
//			t = integrate_uniform_triangle(v1, v2, v3);
//			result += t;
//					
//			v1 = c_dp_t(p[0].x, p[0].y);
//			v2 = c_dp_t(p[4].x, p[4].y);
//			v3 = c_dp_t(p[5].x, p[5].y);
//			t = integrate_uniform_triangle_wall(v1, v2, v3, type);
//			result += t;
			//return result;
			//break;
		}
	case wall_2:
	{
		//double t = 0;
		//double result = 0;
			//// ���� ����������� ��� ������
			//// 1. p2 ������ ������������ (p4,p5,p3) = ������������� �� 3 �������������
			//if (is_point_in_triangle(p[2], p[4], p[5], p[3]))
			//{
			//	c_dp_t v1 = p[5];
			//	c_dp_t v2 = p[3];
			//	c_dp_t v3 = p[4];
			//	//����� �� ����������� � ������� ����������� y ����������
			//	t = integrate_uniform_triangle(v1, v2, v3);
			//	result += t;
			//				
			//	v1 = p[5];
			//	v2 = p[3];
			//	v3 = p[2];
			//	sort_by_y_asc(v1, v2, v3);
			//	t = integrate_uniform_triangle(v1, v2, v3);
			//	result += t;
			//				
			//	v1 = p[3];
			//	v2 = p[2];
			//	v3 = p[4];
			//	sort_by_y_asc(v1, v2, v3);
			//	t = integrate_uniform_triangle(v1, v2, v3);
			//	result += t;
			//}
			//// 2. p3 ������ ������������ (p4,p5,p2) =  ������������� �� 3 �������������
			//else if (is_point_in_triangle(p[3], p[4], p[5], p[2]))
			//{
			//	//����� �� ����������� � ������� ����������� y ����������
			//	c_dp_t v1 = p[5];
			//	c_dp_t v2 = p[2];
			//	c_dp_t v3 = p[4];			
			//	t = integrate_uniform_triangle(v1, v2, v3);
			//	result += t;

			//	v1 = p[5];
			//	v2 = p[2];
			//	v3 = p[3];
			//	sort_by_y_asc(v1, v2, v3);
			//	t = integrate_uniform_triangle(v1, v2, v3);
			//	result += t;

			//	v1 = p[4];
			//	v2 = p[2];
			//	v3 = p[3];
			//	sort_by_y_asc(v1, v2, v3);
			//	t = integrate_uniform_triangle(v1, v2, v3);
			//	result += t;
			//}
			//// 3. �� 1 �� 2 ������� - ���������� ������ 4 ��������, ����������� ��� ������
			//else
			//{
			//	c_dp_t v1 = p[4];
			//	c_dp_t v2 = p[5];
			//	c_dp_t v3 = p[3];
			//	sort_by_y_asc(v1, v2, v3);
			//	t = integrate_uniform_triangle(v1, v2, v3);
			//	result += t;
			//	v1 = p[4];
			//	v2 = p[2];
			//	v3 = p[3];
			//	sort_by_y_asc(v1, v2, v3);
			//	t = integrate_uniform_triangle(v1, v2, v3);
			//	result += t;
			//}
//		if (type == wall_2){
//			c_dp_t v1 = c_dp_t(p[0].x, p[0].y);
//			c_dp_t v2 = c_dp_t(p[1].x, p[1].y);
//			c_dp_t v3 = c_dp_t(p[5].x, p[5].y);
//			sort_by_y_asc(v1, v2, v3);
//			t = integrate_uniform_triangle_wall(v1, v2, v3, type);
//			result += t;
//			v1 = c_dp_t(p[4].x, p[4].y);
//			v2 = c_dp_t(p[1].x, p[1].y);
//			v3 = c_dp_t(p[5].x, p[5].y);
//			sort_by_y_asc(v1, v2, v3);
//			t = integrate_uniform_triangle_wall(v1, v2, v3, type);
//			result += t;
//		}
			//return 0;
			//return result;
		}
	case wall_3_middle_in:
	case wall_3_middle_out:
	case wall_3_middle_at:
		{
			/*double result = 0;
		double t = 0;
		c_dp_t v1 = p[4];
		c_dp_t v2 = p[5];
		c_dp_t v3 = p[3];
		sort_by_y_asc(v1, v2, v3);
		t = integrate_uniform_triangle(a1, b1, c1);
		result += t;
		return result;*/
		}
	case wall_4:
		//return 0;
	case normal:
		{
			double result = 0;
			double t = 0;
			sort_by_y_asc(a1, b1, c1);
			if (i==1 && j == 1)
			{
				flag = 1;
			}
			else
			{
				flag = 0;
			}
			t = integrate_uniform_triangle(prev_dens, a1, b1, c1);
			// if (i==1 && j == 1)
			// {
			// 	printf("%s\n", "integrate");
			// 	printf("%lf\n", t);
			// }
			result += t;
			sort_by_y_asc(a2, b2, c2);
			t = integrate_uniform_triangle(prev_dens, a2, b2, c2);
			result += t;
			return result;
		}
	case concave:
	case convex:
	case pseudo:
		return -1;
	}
	return 0;
}

inline static double get_norm_of_error(double* density, double ts_count_mul_steps)
{
	double r = 0;
	for (int k = 1; k < OY_LEN; ++k)
		for (int j = 1; j < OX_LEN; ++j)
			r += fabs(analytical_solution(ts_count_mul_steps, OX[j], OY[k])
				- density[(OY_LEN + 1) * k + j]);
	return HX * HY * r;
}

inline static void init(double b, double lb, double rb, double bb, double ub,
                        double tau, int time_step_count, int ox_length, int oy_length)
{
	B = b;
	UB = ub;
	BB = bb;
	LB = lb;
	RB = rb;
	TAU = tau;
	TIME_STEP_CNT = time_step_count;
	XY_LEN = (ox_length + 1) * (oy_length + 1);
	OX_LEN = ox_length;
	OX_LEN_1 = ox_length + 1;
	OY_LEN = oy_length;
	OX = new double[OX_LEN_1];
	OY = new double[OY_LEN + 1];
	for (int i = 0; i <= OX_LEN; ++i) OX[i] = lb + i * (rb - lb) / OX_LEN;
	for (int i = 0; i <= OY_LEN; ++i) OY[i] = bb + i * (ub - bb) / OY_LEN;
	HX = OX[1] - OX[0];
	HY = OY[1] - OY[0];
	INVERTED_HX_HY = 1 / HX / HY;
}

inline static void clean()
{
	B = 0;
	UB = 0;
	BB = 0;
	LB = 0;
	RB = 0;
	TAU = 0;
	TIME = 0;
	OX_LEN = 0;
	OY_LEN = 0;
	OX_LEN_1 = 0;
	TIME_STEP_CNT = 0;
	XY_LEN = 0;
	HX = 0;
	HY = 0;
	INVERTED_HX_HY = 0;
	delete [] OX;
	delete [] OY;
}

__global__ void kernel(double* prev_result, double* result)
{
	if (blockIdx.x * blockDim.x + threadIdx.x == 0)
	{
		printf("\nKERNEL PARAMS\n");
		printf("b = %f\n", C_B);
		printf("lbDom = %f\n", C_LB);
	 	printf("rbDom = %f\n", C_RB);
	 	printf("bbDom = %f\n", C_BB);
		printf("ubDom = %f\n", C_UB);
	 	printf("tau = %f\n", C_TAU);
	 	printf("ox length = %d\n", C_OX_LEN + 1);
	 	printf("oy length = %d\n", C_OX_LEN + 1);
	 	printf("PREV_TIME = %lf\n", C_PREV_TIME);
	 	printf("TIME = %lf\n", C_TIME);
	 	printf("C_XY_LEN = %d\n", C_XY_LEN);
	 	printf("%s\n", "OX DEVICE");
	 	for(int i = 0; i < C_OX_LEN + 1; i++)
	 	{
	 		printf("%f ", OX_DEVICE[i]);
	 	}
	 	printf("%s\n", "");
	 	printf("%s\n", "OY DEVICE");
	 	for(int i = 0; i < C_OY_LEN + 1; i++)
	 	{
	 		printf("%f ", OY_DEVICE[i]);
	 	}
	 	printf("%s\n", "");
	 	printf("%s\n", "PREV DENSITY");
	 	for(int i = 0; i < C_OX_LEN + 1; i++)
	 	{
	 		for(int j = 0; j < C_OY_LEN + 1; j++)
		 	{
		 		printf("%f ", prev_result[C_OX_LEN_1 * j + i]);
		 	}
		 	printf("%s\n", "");
	 	}
	 	
	}
	
	 for (int opt = blockIdx.x * blockDim.x + threadIdx.x; opt < C_XY_LEN; opt += blockDim.x * gridDim.x)
	 {		
	 	int i = opt % (C_OX_LEN + 1);
	 	int j = opt / (C_OY_LEN + 1);

	 	// ������ �������
	 	if (j == 0)  // bottom bound
	 	{
	 		result[ opt ]  = 1.1  +  sin( C_TIME * C_HX * j * C_BB );
	 	}
		else if (i == 0) // left bound
		{
			result[ opt ] = 1.1  +  sin( C_TIME * C_HX* i * C_LB );
		}
		else if (j == C_OY_LEN) // upper bound
		{ 
			result[ opt ] = 1.1  +  sin( C_TIME * C_HX * i * C_UB );
		}
		else if (i == C_OX_LEN) // right bound
		{ 
			result[ opt ] = 1.1  +  sin(  C_TIME * C_HX * j * C_RB );
		}
		else if (i > 0 && j > 0 && j != C_OY_LEN && i != C_OX_LEN)
		{                   
			double t = integrate(prev_result, i, j);	
			result[ opt ] =  t * C_INVERTED_HX_HY;
			if (opt == 13) 
			{
				printf("%s\n", "result = 13");
				printf("%lf\n", t);
				printf("%lf\n", result[ opt ]);
			}
			result[ opt ] += C_TAU * func_f(C_B, C_TIME, C_UB, C_BB, C_LB, C_RB, OX_DEVICE[i], OY_DEVICE[j]);
			if (opt == 13) 
			{
				printf("%s\n", " F result = 13");
				printf("%lf\n", C_TAU);
				printf("%lf\n", result[ opt ] );
				printf("%lf\n", func_f(C_B, C_TIME, C_UB, C_BB, C_LB, C_RB, OX_DEVICE[i], OY_DEVICE[j]) );
			}
		}
	 }
}

float solve_cuda(double* density)
{
//	const int gridSize = 256;
//	const int blockSize =  512; 
	const int gridSize = 1;
	const int blockSize =  1;
	double *result = NULL, *prev_result = NULL, *ox = NULL, *oy=NULL;
	int size = sizeof(double)*XY_LEN;
	double *prev_result_h = new double[XY_LEN];
	for (int j = 0; j < OY_LEN + 1; j++)
	{
		for (int i = 0; i < OX_LEN_1; i++)
		{
			prev_result_h[OX_LEN_1 * j + i] = analytical_solution(0, OX[i], OY[j]);
		}
	}

	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_TAU), &TAU, sizeof(double)));	
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_B), &B, sizeof(double)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_LB), &LB, sizeof(double)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_RB), &RB, sizeof(double)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_BB), &BB, sizeof(double)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_UB), &UB, sizeof(double)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_INVERTED_HX_HY), &INVERTED_HX_HY, sizeof(double)));	
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_HX), &HX, sizeof(int)));	
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_HY), &HY, sizeof(int)));	
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_OX_LEN_1), &OX_LEN_1, sizeof(int)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_XY_LEN), &XY_LEN, sizeof(int)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_OX_LEN), &OX_LEN, sizeof(int)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_OY_LEN), &OY_LEN, sizeof(int)));
	
	checkCuda(hipMalloc((void**)&(result), size) );
	checkCuda(hipMemset(result, 0, size) );
	checkCuda(hipMalloc((void**)&(prev_result), size) );
	checkCuda(hipMalloc((void**)&(ox), sizeof(ox)*(OX_LEN+1)));
	checkCuda(hipMalloc((void**)&(oy), sizeof(oy)*(OY_LEN+1)));
	checkCuda(hipMemcpy(ox, OX, sizeof(ox)*(OX_LEN + 1), hipMemcpyHostToDevice));	
	checkCuda(hipMemcpy(oy, OY, sizeof(oy)*(OY_LEN + 1), hipMemcpyHostToDevice));	
	checkCuda(hipMemcpy(prev_result, prev_result_h, size, hipMemcpyHostToDevice));	
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(OX_DEVICE), &ox, sizeof(ox)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(OY_DEVICE), &oy, sizeof(oy)));	

	hipEventRecord(start, 0);   

	TIME = 0;
	int tl = 0;
	TIME_STEP_CNT = 2;
	int tempTl  = TIME_STEP_CNT - 1;
	while(tl < tempTl)
	{
		checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_PREV_TIME), &TIME, sizeof(double)));
     	TIME = TAU * (tl+1);
	    checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_TIME), &TIME, sizeof(double)));	
	    kernel<<<gridSize, blockSize>>>(prev_result, result);
		// checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_PREV_TIME), &TIME, sizeof(double)));
  		// TIME = TAU * (tl+2);
		// checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(C_TIME), &TIME, sizeof(double)));	
		// kernel<<<gridSize, blockSize>>>(result, prev_result); 
	    tl += 2;            
	}

	printf("%s\n", "");
	//checkCuda(hipMemcpy(density, prev_result, size, hipMemcpyDeviceToHost));	
	checkCuda(hipMemcpy(density, result, size, hipMemcpyDeviceToHost));	
	for (int j = 0; j < OX_LEN_1; j++)
	{
		for (int i = 0; i < OY_LEN + 1; i++)
		{
			printf("%lf ", density[OX_LEN_1 * i + j]);			
		}
		printf("%s\n", "");
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("Computation Time %f\n", time);
	hipFree(result);
	hipFree(prev_result);
	hipFree(ox);
	hipFree(oy);
	hipDeviceReset();
	delete[] prev_result_h;
	return time;
}

inline void print_matrix11(double* a, int n, int m, int precision = 8) {
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < m; ++j) {
			int k = i * n + j;
			switch (precision) {
			case 1:
				printf("%.1f ", a[k]);
				break;
			case 2:
				printf("%.2f ", a[k]);
				break;
			case 3:
				printf("%.3f ", a[k]);
				break;
			case 4:
				printf("%.4f ", a[k]);
				break;
			case 5:
				printf("%.5f ", a[k]);
				break;
			case 6:
				printf("%.6f ", a[k]);
				break;
			case 7:
				printf("%.7f ", a[k]);
				break;
			case 8:
				printf("%.8f ", a[k]);
				break;
			}
		}
		printf("\n");
	}
}

double* compute_density_cuda_internal(double b, double lb, double rb, double bb, double ub,
                        double tau, int time_step_count, int ox_length, int oy_length, double& norm, float& time)
{
#ifdef __NVCC__	
    init(b, lb, rb, bb, ub, tau, time_step_count, ox_length, oy_length);
	double* density = new double[XY_LEN];
	print_params(B, LB, RB, BB, UB, TAU, TIME_STEP_CNT, OX_LEN, OY_LEN);
	time = solve_cuda(density);
	//print_matrix11(density, 11, 11);
	norm = get_norm_of_error(density, TIME_STEP_CNT * TAU);
	clean();
	return density;
#else
        return NULL;
#endif
}
